#include "SU.hh"

#include "scuda.h"
#include "squad.h"

#include <thrust/device_ptr.h>
#include <thrust/copy.h>


template<typename T>
T* SU::upload(const T* h, unsigned num_items )
{
    T* d ;
    hipMalloc(&d, num_items*sizeof(T));
    hipMemcpy(d, h, num_items*sizeof(T), hipMemcpyHostToDevice);
    return d ; 
}



/**
SU::select_copy_device_to_host
-------------------------------

1. apply thrust::count_if to *d* with *selector* functor yielding *num_select* 
2. allocate *d_select* with num_select*sizeof(T) bytes
3. thrust::copy_if from *d* to *d_select* using the *selector* functor
4. host new T[num_select] allocation 
5. copies from *d_select* to the *num_select* host array *h* using the selector 

This API is awkward because the number selected is not known when making the call.
For example it would be difficult to populate an NP array using this without 
making copies. 

**/

template<typename T>
void SU::select_copy_device_to_host( T** h, unsigned& num_select,  T* d, unsigned num_d, const qselector<T>& selector  )
{   
    thrust::device_ptr<T> td(d);
    num_select = thrust::count_if(td, td+num_d , selector );
    std::cout << " num_select " << num_select << std::endl ;
    
    T* d_select ;   
    hipMalloc(&d_select,     num_select*sizeof(T));
    //hipMemset(d_select, 0,   num_select*sizeof(T));
    thrust::device_ptr<T> td_select(d_select);
    
    thrust::copy_if(td, td+num_d , td_select, selector );
    
    *h = new T[num_select] ; 
    hipMemcpy(*h, d_select, num_select*sizeof(T), hipMemcpyDeviceToHost);
}


/**
SU::select_count
------------------

1. apply thrust::count_if to *d* with *selector* functor yielding *num_select* 

**/

template<typename T>
unsigned SU::select_count( T* d, unsigned num_d,  qselector<T>& selector )
{
    thrust::device_ptr<T> td(d);
    return thrust::count_if(td, td+num_d , selector );
}


/**
SU::select_copy_device_to_host_presized
-----------------------------------------

The host array must be presized to fit the selection, determine the size using *select_count* with the same selector. 

1. allocates *d_select* with num_select*sizeof(T) bytes
2. thrust::copy_if from *d* to *d_select* using the selector functor
3. copies from *d_select* to the *num_select* presized host array *h* using the selector

**/

template<typename T>
void SU::select_copy_device_to_host_presized( T* h, T* d, unsigned num_d, const qselector<T>& selector, unsigned num_select  )
{
    thrust::device_ptr<T> td(d);

    T* d_select ;
    hipMalloc(&d_select,     num_select*sizeof(T));
    //hipMemset(d_select, 0,   num_select*sizeof(T));
    thrust::device_ptr<T> td_select(d_select);

    thrust::copy_if(td, td+num_d , td_select, selector );

    hipMemcpy(h, d_select, num_select*sizeof(T), hipMemcpyDeviceToHost);
}

template SYSRAP_API unsigned SU::select_count( quad4* , unsigned, qselector<quad4>& ); 
template SYSRAP_API quad4*   SU::upload(const quad4* , unsigned ); 
template SYSRAP_API void     SU::select_copy_device_to_host( quad4** h, unsigned& ,  quad4* , unsigned , const qselector<quad4>&  ); 
template SYSRAP_API void     SU::select_copy_device_to_host_presized( quad4*, quad4*, unsigned, const qselector<quad4>& , unsigned ); 

 




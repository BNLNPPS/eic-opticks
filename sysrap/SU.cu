#include "SU.hh"

#include "scuda.h"
#include "squad.h"

#include <thrust/device_ptr.h>
#include <thrust/copy.h>


template<typename T>
T* SU::upload(const T* h, unsigned num_items )
{
    T* d ;
    hipMalloc(&d, num_items*sizeof(T));
    hipMemcpy(d, h, num_items*sizeof(T), hipMemcpyHostToDevice);
    return d ; 
}

template SYSRAP_API quad4*   SU::upload(const quad4* , unsigned ); 


/**
SU::deprecated_select_copy_device_to_host
--------------------------------------------

1. apply thrust::count_if to *d* with *selector* functor yielding *num_select* 
2. allocate *d_select* with num_select*sizeof(T) bytes
3. thrust::copy_if from *d* to *d_select* using the *selector* functor
4. host new T[num_select] allocation 
5. copies from *d_select* to the *num_select* host array *h* using the selector 

This API is deprecated because its awkward as the number selected is not known when making the call.
For example it would be difficult to populate an NP array using this without 
making copies. 

**/

template<typename T>
void SU::deprecated_select_copy_device_to_host( T** h, unsigned& num_select,  T* d, unsigned num_d, const qselector<T>& selector  )
{   
    thrust::device_ptr<T> td(d);
    num_select = thrust::count_if(td, td+num_d , selector );
    std::cout << " num_select " << num_select << std::endl ;
    
    T* d_select ;   
    hipMalloc(&d_select,     num_select*sizeof(T));
    thrust::device_ptr<T> td_select(d_select);
    
    thrust::copy_if(td, td+num_d , td_select, selector );
    
    *h = new T[num_select] ; 
    hipMemcpy(*h, d_select, num_select*sizeof(T), hipMemcpyDeviceToHost);
}
template SYSRAP_API void     SU::deprecated_select_copy_device_to_host( quad4** h, unsigned& ,  quad4* , unsigned , const qselector<quad4>&  ); 


/**
SU::count_if
------------------

1. apply thrust::count_if to *d* with *selector* functor yielding *num_select* 

**/

template<typename T>
unsigned SU::count_if( const T* d, unsigned num_d,  qselector<T>& selector )
{
    thrust::device_ptr<const T> td(d);
    return thrust::count_if(td, td+num_d , selector );
}

template SYSRAP_API unsigned SU::count_if( const quad4* , unsigned, qselector<quad4>& ); 


/**
SU::device_alloc
-------------------

1. allocates *d* with num*sizeof(T) bytes

**/

template<typename T>
T* SU::device_alloc( unsigned num  )
{
    T* d ;
    hipMalloc(&d,  num*sizeof(T));
    return d ; 
}
template SYSRAP_API quad4* SU::device_alloc( unsigned ); 


template<typename T>
void SU::device_zero( T* d, unsigned num )
{
    hipMemset(d, 0,  num*sizeof(T));
}
template SYSRAP_API void SU::device_zero( quad4*, unsigned ); 


/**
SU::copy_if_device_to_device_presized
-----------------------------------------

The d_select array must be presized to fit the selection, determine the size using *select_count* with the same selector. 

2. thrust::copy_if from *d* to *d_select* using the selector functor
3. copies from *d_select* to the *num_select* presized host array *h* using the selector

**/

template<typename T>
void SU::copy_if_device_to_device_presized( T* d_select, const T* d, unsigned num_d, const qselector<T>& selector )
{
    thrust::device_ptr<const T> td(d);
    thrust::device_ptr<T> td_select(d_select);
    thrust::copy_if(td, td+num_d , td_select, selector );
}

template SYSRAP_API void SU::copy_if_device_to_device_presized( quad4*, const quad4*, unsigned, const qselector<quad4>& ); 


template<typename T>
void SU::copy_device_to_host_presized( T* h, const T* d, unsigned num  )
{
    hipMemcpy(h, d, num*sizeof(T), hipMemcpyDeviceToHost);
}
template SYSRAP_API void SU::copy_device_to_host_presized( quad4*, const quad4*, unsigned ); 




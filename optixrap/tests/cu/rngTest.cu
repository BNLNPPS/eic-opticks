#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <optix_world.h>

using namespace optix;

//  rng_states rng_skipahead
#include "ORng.hh"

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

rtBuffer<float>  out_buffer;

RT_PROGRAM void rngTest()
{
    unsigned long long photon_id = launch_index.x ;
    hiprandState rng = rng_states[photon_id];
    float u = hiprand_uniform(&rng);  
    out_buffer[photon_id] = u ; 
}

RT_PROGRAM void exception()
{
    rtPrintExceptionDetails();
}


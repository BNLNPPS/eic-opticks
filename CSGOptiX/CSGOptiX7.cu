#include "hip/hip_runtime.h"
/**
CSGOptiX7.cu 
===================

NB: ONLY CODE THAT MUST BE HERE DUE TO OPTIX DEPENDENCY SHOULD BE HERE
everything else should be located elsewhere : mostly in qudarap: sevent, qsim 
or the sysrap basis types sphoton quad4 quad2 etc.. where the code is reusable 
and more easily tested. 

**/

#include <optix.h>

#include "SRG.h"
#include "scuda.h"
#include "squad.h"
#include "sqat4.h"
#include "sphoton.h"
#include "scerenkov.h"

// simulation 
#include <hiprand/hiprand_kernel.h>

#include "qstate.h"
#include "qsim.h"
#include "sevent.h"

#include "csg_intersect_leaf.h"
#include "csg_intersect_node.h"
#include "csg_intersect_tree.h"

#include "Binding.h"
#include "Params.h"

#ifdef WITH_PRD
#include "Pointer.h"
#endif

extern "C" { __constant__ Params params ;  }

/**
trace : pure function, with no use of params, everything via args
-------------------------------------------------------------------

Outcome of trace is to populate *prd* by payload and attribute passing.
When WITH_PRD macro is defined only 2 32-bit payload values are used to 
pass the 64-bit  pointer, otherwise more payload and attributes values 
are used to pass the contents IS->CH->RG. 

See __closesthit__ch to see where the payload p0-p5 comes from.
**/

static __forceinline__ __device__ void trace(
        OptixTraversableHandle handle,
        float3                 ray_origin,
        float3                 ray_direction,
        float                  tmin,
        float                  tmax,
        quad2*                 prd
        )   
{
    const float rayTime = 0.0f ; 
    OptixVisibilityMask visibilityMask = 1u  ; 
    OptixRayFlags rayFlags = OPTIX_RAY_FLAG_DISABLE_ANYHIT ;   // OPTIX_RAY_FLAG_NONE 
    const unsigned SBToffset = 0u ; 
    const unsigned SBTstride = 1u ; 
    const unsigned missSBTIndex = 0u ; 
#ifdef WITH_PRD
    uint32_t p0, p1 ; 
    packPointer( prd, p0, p1 ); 
    optixTrace(
            handle,
            ray_origin,
            ray_direction,
            tmin,
            tmax,
            rayTime,
            visibilityMask,
            rayFlags,
            SBToffset,
            SBTstride,
            missSBTIndex,
            p0, p1
            );
#else
    uint32_t p0, p1, p2, p3, p4, p5, p6, p7  ; 
    optixTrace(
            handle,
            ray_origin,
            ray_direction,
            tmin,
            tmax,
            rayTime,
            visibilityMask,
            rayFlags,
            SBToffset,
            SBTstride,
            missSBTIndex,
            p0, p1, p2, p3, p4, p5, p6, p7
            );
    // unclear where the uint_as_float CUDA device function is defined, seems CUDA intrinsic without header ?
    prd->q0.f.x = uint_as_float( p0 );
    prd->q0.f.y = uint_as_float( p1 );
    prd->q0.f.z = uint_as_float( p2 );
    prd->q0.f.w = uint_as_float( p3 ); 
    prd->set_identity(p4) ; 
    prd->set_boundary(p5) ;  
    prd->set_lposcost(uint_as_float(p6)) ;  
    prd->set_iindex(p7) ;  
#endif
}


__forceinline__ __device__ uchar4 make_color( const float3& normal, unsigned identity, unsigned boundary )  // pure 
{
    float scale = 1.f ; 
    return make_uchar4(
            static_cast<uint8_t>( clamp( normal.x, 0.0f, 1.0f ) *255.0f )*scale ,
            static_cast<uint8_t>( clamp( normal.y, 0.0f, 1.0f ) *255.0f )*scale ,
            static_cast<uint8_t>( clamp( normal.z, 0.0f, 1.0f ) *255.0f )*scale ,
            255u
            );
}

/**
render : non-pure, uses params for viewpoint inputs and pixels output 
-----------------------------------------------------------------------

**/

static __forceinline__ __device__ void render( const uint3& idx, const uint3& dim, quad2* prd )
{
    float2 d = 2.0f * make_float2(
            static_cast<float>(idx.x)/static_cast<float>(dim.x),
            static_cast<float>(idx.y)/static_cast<float>(dim.y)
            ) - 1.0f;

    const bool yflip = true ;
    if(yflip) d.y = -d.y ;

    const unsigned cameratype = params.cameratype ;  
    const float3 dxyUV = d.x * params.U + d.y * params.V ; 
    const float3 origin    = cameratype == 0u ? params.eye                     : params.eye + dxyUV    ;
    const float3 direction = cameratype == 0u ? normalize( dxyUV + params.W )  : normalize( params.W ) ;
    //                           cameratype 0u:perspective,                    1u:orthographic

    trace( 
        params.handle,
        origin,
        direction,
        params.tmin,
        params.tmax,
        prd
    );

    float3 position = origin + direction*prd->distance() ;
    const float3* normal = prd->normal();  
    float3 diddled_normal = normalize(*normal)*0.5f + 0.5f ; // diddling lightens the render, with mid-grey "pedestal" 
    unsigned index = idx.y * params.width + idx.x ;

    params.pixels[index] = make_color( diddled_normal, prd->identity(), prd->boundary() ); 
    params.isect[index]  = make_float4( position.x, position.y, position.z, uint_as_float(prd->identity())) ; 
}
 
/**
simulate : uses params for input: gensteps, seeds and output photons 
----------------------------------------------------------------------

Contrast with the monolithic old way with OptiXRap/cu/generate.cu:generate 

This method aims to get as much as possible of its functionality from 
separately implemented and tested headers. 

The big thing that CSGOptiX provides is geometry intersection, only that must be here. 
Everything else should be implemented and tested elsewhere, mostly in QUDARap headers.

Hence this "simulate" needs to act as a coordinator. 
Params take central role in enabling this:


Params
~~~~~~~

* CPU side params including qsim.h sevent.h pointers instanciated in CSGOptiX::CSGOptiX 
  and populated by CSGOptiX::init methods before being uploaded by CSGOptiX::prepareParam 


COMPARE WITH qsim::mock_propagate

**/

static __forceinline__ __device__ void simulate( const uint3& launch_idx, const uint3& dim, quad2* prd )
{
    sevent* evt      = params.evt ; 
    if (launch_idx.x >= evt->num_photon) return;

    unsigned idx = launch_idx.x ;  // aka photon_id
    unsigned genstep_id = evt->seed[idx] ; 
    const quad6& gs     = evt->genstep[genstep_id] ; 
     
    qsim* sim = params.sim ; 
    hiprandState rng = sim->rngstate[idx] ;    // TODO: skipahead using an event_id 

    sphoton p = {} ;   
    srec rec = {} ; 
    sseq seq = {} ;  // seqhis..

    sim->generate_photon(p, rng, gs, idx, genstep_id );  

    qstate state = {} ; 

    int command = START ; 
    int bounce = 0 ;  
    while( bounce < evt->max_bounce )
    {    
        trace( 
            params.handle,
            p.pos,
            p.mom,
            params.tmin,
            params.tmax,
            prd
        );        // trace populates prd with geometry info : intersect normal, distance, identity

        if(evt->record) evt->record[evt->max_record*idx+bounce] = p ;  
        if(evt->rec) evt->add_rec( rec, idx, bounce, p ); 
        if(evt->seq) seq.add_nibble( bounce, p.flag(), p.boundary() ); 
        if(evt->prd) evt->prd[evt->max_prd*idx+bounce] = *prd ; 


        //printf("//OptiX7Test.cu:simulate idx %d bounce %d boundary %d \n", idx, bounce, prd->boundary() ); 
        if( prd->boundary() == 0xffffu ) break ;   // propagate can do nothing meaningful without a boundary 

        command = sim->propagate(bounce, p, state, prd, rng, idx ); 
        bounce++;     
        if(command == BREAK) break ;    
    }    

    if( evt->record && bounce < evt->max_record ) evt->record[evt->max_record*idx+bounce] = p ;  
    if( evt->rec    && bounce < evt->max_rec    ) evt->add_rec(rec, idx, bounce, p ); 
    if( evt->seq    && bounce < evt->max_seq    ) seq.add_nibble(bounce, p.flag(), p.boundary() );

    evt->photon[idx] = p ; 
    if(evt->seq) evt->seq[idx] = seq ;
}

/**
simtrace
----------

Used for making 2D cross section views of geometry intersects  

Note how seeding is still needed here despite the highly artificial 
nature of the center-extent grid of gensteps as the threads of the launch 
still needs to access different gensteps across the grid. 

TODO: Compose frames of pixels, isect and "fphoton" within the cegs window
using the positions of the intersect "photons".
Note that multiple threads may be writing to the same pixel 
hat is apparently not a problem, just which does it is uncontrolled.

unsigned index = iz * params.width + ix ;
if( index > 0 )
{
    params.pixels[index] = make_uchar4( 255u, 0u, 0u, 255u) ;
    params.isect[index] = make_float4( ipos.x, ipos.y, ipos.z, uint_as_float(identity)) ; 
    params.fphoton[index] = p ; 
}
**/

static __forceinline__ __device__ void simtrace( const uint3& launch_idx, const uint3& dim, quad2* prd )
{
    unsigned idx = launch_idx.x ;  // aka photon_id
    sevent* evt  = params.evt ; 
    if (idx >= evt->num_simtrace) return;

    unsigned genstep_id = evt->seed[idx] ; 
    if(idx == 0) printf("//OptiX7Test.cu:simtrace idx %d genstep_id %d \n", idx, genstep_id ); 

    const quad6& gs     = evt->genstep[genstep_id] ; 
     
    qsim* sim = params.sim ; 
    hiprandState rng = sim->rngstate[idx] ;   

    quad4 p ;  
    sim->generate_photon_simtrace(p, rng, gs, idx, genstep_id );  

    const float3& pos = (const float3&)p.q0.f  ; 
    const float3& mom = (const float3&)p.q1.f ; 

    trace( 
        params.handle,
        pos,
        mom,
        params.tmin,
        params.tmax,
        prd
    );

    evt->add_simtrace( idx, p, prd, params.tmin ); 

}

/**
for angular efficiency need intersection point in object frame to get the angles  
**/

extern "C" __global__ void __raygen__rg()
{
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    quad2 prd ; 
    prd.zero(); 
  
    switch( params.raygenmode )
    {
        case SRG_RENDER:    render(   idx, dim, &prd ) ; break ;  
        case SRG_SIMTRACE:  simtrace( idx, dim, &prd ) ; break ;  
        case SRG_SIMULATE:  simulate( idx, dim, &prd ) ; break ;  
    }
} 


#ifdef WITH_PRD
#else
/**
*setPayload* is used from __closesthit__ and __miss__ providing communication to __raygen__ optixTrace call
**/
static __forceinline__ __device__ void setPayload( float normal_x, float normal_y, float normal_z, float distance, unsigned identity, unsigned boundary, float lposcost, unsigned iindex )
{
    optixSetPayload_0( float_as_uint( normal_x ) );
    optixSetPayload_1( float_as_uint( normal_y ) );
    optixSetPayload_2( float_as_uint( normal_z ) );
    optixSetPayload_3( float_as_uint( distance ) );
    optixSetPayload_4( identity );
    optixSetPayload_5( boundary );
    optixSetPayload_6( lposcost );  
    optixSetPayload_7( iindex   );  

    // num_payload_values PIP::PIP must match the payload slots used up to maximum of 8 
    // NB : payload is distinct from attributes
}
#endif

/**
__miss__ms
-------------

* missing "normal" is somewhat render specific and this is used for 
  all raygenmode but Miss should never happen with real simulations 
* Miss can happen with simple geometry testing however

**/


extern "C" __global__ void __miss__ms()
{
    MissData* ms  = reinterpret_cast<MissData*>( optixGetSbtDataPointer() );
    const unsigned identity = 0xffffffffu ; 
    const unsigned boundary = 0xffffu ;
    const float lposcost = 0.f ; 
  
#ifdef WITH_PRD
    quad2* prd = getPRD<quad2>(); 

    prd->q0.f.x = ms->r ;   
    prd->q0.f.y = ms->g ; 
    prd->q0.f.z = ms->b ; 
    prd->q0.f.w = 0.f ; 

    prd->q1.u.x = 0u ; 
    prd->q1.u.y = 0u ; 
    prd->q1.u.z = 0u ; 
    prd->q1.u.w = 0u ; 

    prd->set_boundary(boundary); 
    prd->set_identity(identity); 
    prd->set_lposcost(lposcost); 
#else
    setPayload( ms->r, ms->g, ms->b, 0.f, identity, boundary, lposcost );  // communicate from ms->rg
#endif
}

/**
__closesthit__ch : pass attributes from __intersection__ into setPayload
============================================================================

optixGetInstanceId 
    flat instance_idx over all transforms in the single IAS, 
    JUNO maximum ~50,000 (fits with 0xffff = 65535)

optixGetPrimitiveIndex
    local index of AABB within the GAS, 
    instanced solids adds little to the number of AABB, 
    most come from unfortunate repeated usage of prims in the non-instanced global
    GAS with repeatIdx 0 (JUNO up to ~4000)

optixGetRayTmax
    In intersection and CH returns the current smallest reported hitT or the tmax passed into rtTrace 
    if no hit has been reported


**/

extern "C" __global__ void __closesthit__ch()
{
    unsigned iindex = optixGetInstanceIndex() ;    // 0-based index within IAS
    unsigned instance_id = optixGetInstanceId() ;  // user supplied instanceId, see IAS_Builder::Build and InstanceId.h 
    unsigned prim_idx = optixGetPrimitiveIndex() ; // GAS_Builder::MakeCustomPrimitivesBI_11N  (1+index-of-CSGPrim within CSGSolid/GAS)
    unsigned identity = (( prim_idx & 0xffff ) << 16 ) | ( instance_id & 0xffff ) ; 

#ifdef WITH_PRD
    quad2* prd = getPRD<quad2>(); 

    prd->set_identity( identity ) ;
    prd->set_iindex(   iindex ) ;
    //printf("//__closesthit__ch prd.boundary %d \n", prd->boundary() );  // boundary set in IS for WITH_PRD
    float3* normal = prd->normal(); 
    *normal = optixTransformNormalFromObjectToWorldSpace( *normal ) ;  

#else
    const float3 local_normal =    // geometry object frame normal at intersection point 
        make_float3(
                uint_as_float( optixGetAttribute_0() ),
                uint_as_float( optixGetAttribute_1() ),
                uint_as_float( optixGetAttribute_2() )
                );

    const float distance = uint_as_float(  optixGetAttribute_3() ) ;  
    unsigned boundary = optixGetAttribute_4() ; 
    const float lposcost = uint_as_float( optixGetAttribute_5() ) ; 
    float3 normal = optixTransformNormalFromObjectToWorldSpace( local_normal ) ;  

    setPayload( normal.x, normal.y, normal.z, distance, identity, boundary, lposcost, iindex );  // communicate from ch->rg
#endif
}

/**
__intersection__is
----------------------

HitGroupData provides the numNode and nodeOffset of the intersected CSGPrim.
Which Prim gets intersected relies on the CSGPrim::setSbtIndexOffset

Note that optixReportIntersection returns a bool, but that is 
only relevant when using anyHit as it provides a way to ignore hits.
But Opticks does not used any anyHit so the returned bool should 
always be true. 

The attributes passed into optixReportIntersection are 
available within the CH (and AH) programs. 

**/

extern "C" __global__ void __intersection__is()
{
    HitGroupData* hg  = (HitGroupData*)optixGetSbtDataPointer();  
    int nodeOffset = hg->nodeOffset ; 

    const CSGNode* node = params.node + nodeOffset ;  // root of tree
    const float4* plan = params.plan ;  
    const qat4*   itra = params.itra ;  

    const float  t_min = optixGetRayTmin() ; 
    const float3 ray_origin = optixGetObjectRayOrigin();
    const float3 ray_direction = optixGetObjectRayDirection();

    float4 isect ; // .xyz normal .w distance 
    if(intersect_prim(isect, node, plan, itra, t_min , ray_origin, ray_direction ))  
    {
        const float lposcost = normalize_z(ray_origin + isect.w*ray_direction ) ;  
        const unsigned hitKind = 0u ;            // only 8bit : could use to customize how attributes interpreted
        const unsigned boundary = node->boundary() ;  // all nodes of tree have same boundary 
        //printf("//__intersection__is boundary %d \n", boundary ); 

#ifdef WITH_PRD
        if(optixReportIntersection( isect.w, hitKind))
        {
            quad2* prd = getPRD<quad2>(); 
            prd->q0.f = isect ;  // .w:distance and .xyz:normal which starts as the local frame one 
            prd->set_boundary(boundary) ; 
            prd->set_lposcost(lposcost); 
            //printf("//__intersection__is prd.set_boundary %d \n", boundary ); 
        }   
#else
        unsigned a0, a1, a2, a3, a4, a5  ; // MUST CORRESPOND TO num_attribute_values in PIP::PIP 
        a0 = float_as_uint( isect.x );     // isect.xyz is object frame normal of geometry at intersection point 
        a1 = float_as_uint( isect.y );
        a2 = float_as_uint( isect.z );
        a3 = float_as_uint( isect.w ) ; 
        a4 = boundary ; 
        a5 = float_as_uint( lposcost ); 
        optixReportIntersection( isect.w, hitKind, a0, a1, a2, a3, a4, a5 );   
#endif
        // IS:optixReportIntersection writes the attributes that can be read in CH and AH programs 
        // max 8 attribute registers, see PIP::PIP, communicate to __closesthit__ch 
    }
}
// story begins with intersection

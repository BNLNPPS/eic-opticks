#include "hip/hip_runtime.h"
/**

REMEMBER TO KEEP CODE HERE TO A MINIMUM : PUT AS MUCH AS POSSIBLE INTO THE 
MORE EASILY TESTED FROM MULTIPLE ENVIRONMENTS HEADERS 

**/

#include "stdio.h"
#include "hiprand/hiprand_kernel.h"
#include "scuda.h"
#include "qgs.h"
#include "qctx.h"


__global__ void _QCtx_rng_sequence_0(qctx* ctx, float* rs, unsigned num_items )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= num_items) return;
    hiprandState rng = *(ctx->r + id) ; 
    float u = hiprand_uniform(&rng) ;
    if(id % 100000 == 0) printf("//_QCtx_rng_sequence id %d u %10.4f    \n", id, u  ); 
    rs[id] = u ; 
}

extern "C" void QCtx_rng_sequence_0(dim3 numBlocks, dim3 threadsPerBlock, qctx* ctx, float* rs, unsigned num_items )
{
    printf("//QCtx_rng_sequence_0 num_items %d \n", num_items ); 
    _QCtx_rng_sequence_0<<<numBlocks,threadsPerBlock>>>( ctx, rs, num_items );
} 






__global__ void _QCtx_rng_sequence_f(qctx* ctx, float* seq, unsigned ni, unsigned nv, unsigned ioffset )
{
    unsigned i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= ni) return;
    hiprandState rng = *(ctx->r + i + ioffset) ; 
    unsigned ibase = i*nv ; 

    for(unsigned v=0 ; v < nv ; v++)
    {
        float u = hiprand_uniform(&rng) ;
        seq[ibase+v] = u ;
    } 
}
extern "C" void QCtx_rng_sequence_f(dim3 numBlocks, dim3 threadsPerBlock, qctx* ctx, float*  seq, unsigned ni, unsigned nv, unsigned ioffset )
{
    printf("//QCtx_rng_sequence_f ni %d nv %d ioffset %d  \n", ni, nv, ioffset ); 
    _QCtx_rng_sequence_f<<<numBlocks,threadsPerBlock>>>( ctx, seq, ni, nv, ioffset );

}

__global__ void _QCtx_rng_sequence_d(qctx* ctx, double* seq, unsigned ni, unsigned nv, unsigned ioffset )
{
    unsigned i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= ni) return;
    hiprandState rng = *(ctx->r + i + ioffset) ; 
    unsigned ibase = i*nv ; 

    for(unsigned v=0 ; v < nv ; v++)
    {
        double u = hiprand_uniform_double(&rng) ;
        seq[ibase+v] = u ;
    } 
}
extern "C" void QCtx_rng_sequence_d(dim3 numBlocks, dim3 threadsPerBlock, qctx* ctx, double* seq, unsigned ni, unsigned nv, unsigned ioffset )
{
    printf("//QCtx_rng_sequence_d ni %d nv %d ioffset %d  \n", ni, nv, ioffset ); 
    _QCtx_rng_sequence_d<<<numBlocks,threadsPerBlock>>>( ctx, seq, ni, nv, ioffset );
}





/**
HMM hd_factor is more appropriate as a property of the uploaded texture than it is an input argument 
TODO: rearrange hd_factor 
**/

__global__ void _QCtx_generate_scint_wavelength(qctx* ctx, float* wavelength, unsigned num_wavelength, unsigned hd_factor )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= num_wavelength) return;

    hiprandState rng = *(ctx->r + id) ; 

    float wl ; 
    switch(hd_factor)
    {
        case 0:  wl = ctx->scint_wavelength_hd0(rng)  ; break ; 
        case 10: wl = ctx->scint_wavelength_hd10(rng) ; break ; 
        case 20: wl = ctx->scint_wavelength_hd20(rng) ; break ; 
        default: wl = 0.f ; 
    }
    if(id % 100000 == 0) printf("//_QCtx_generate_scint_wavelength id %d hd_factor %d wl %10.4f    \n", id, hd_factor, wl  ); 
    wavelength[id] = wl ; 
}

extern "C" void QCtx_generate_scint_wavelength(dim3 numBlocks, dim3 threadsPerBlock, qctx* ctx, float* wavelength, unsigned num_wavelength, unsigned hd_factor ) 
{
    printf("//QCtx_generate_scint_wavelength num_wavelength %d \n", num_wavelength ); 
    _QCtx_generate_scint_wavelength<<<numBlocks,threadsPerBlock>>>( ctx, wavelength, num_wavelength, hd_factor );
} 

/**
genstep provisioning ? gensteps need to be uploaded with pointer held in qctx 
but for testing need to be able to manually fabricate a genstep
**/

__global__ void _QCtx_generate_cerenkov_wavelength(qctx* ctx, float* wavelength, unsigned num_wavelength )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= num_wavelength) return;

    hiprandState rng = *(ctx->r + id) ; 

    float wl = ctx->cerenkov_wavelength(id, rng);   

    if(id % 100000 == 0) printf("//_QCtx_generate_cerenkov_wavelength id %d wl %10.4f    \n", id, wl  ); 
    wavelength[id] = wl ; 
}


extern "C" void QCtx_generate_cerenkov_wavelength(dim3 numBlocks, dim3 threadsPerBlock, qctx* ctx, float* wavelength, unsigned num_wavelength ) 
{
    printf("//QCtx_generate_cerenkov_wavelength num_wavelength %d \n", num_wavelength ); 
    _QCtx_generate_cerenkov_wavelength<<<numBlocks,threadsPerBlock>>>( ctx, wavelength, num_wavelength );
} 





__global__ void _QCtx_generate_cerenkov_photon(qctx* ctx, quad4* photon, unsigned num_photon, int print_id )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= num_photon) return;

    hiprandState rng = *(ctx->r + id) ; 

    quad4 p ;   
    ctx->cerenkov_photon(p, id, rng, print_id);   

    if(id % 100000 == 0) printf("//_QCtx_generate_cerenkov_photon id %d \n", id  ); 
    photon[id] = p ; 
}

extern "C" void QCtx_generate_cerenkov_photon(dim3 numBlocks, dim3 threadsPerBlock, qctx* ctx, quad4* photon, unsigned num_photon, int print_id ) 
{
    printf("//QCtx_generate_cerenkov_photon num_photon %d \n", num_photon ); 
    _QCtx_generate_cerenkov_photon<<<numBlocks,threadsPerBlock>>>( ctx, photon, num_photon, print_id );
} 








__global__ void _QCtx_generate_photon(qctx* ctx, quad4* photon, unsigned num_photon )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= num_photon) return;
    
    //ctx->r += id ;   
    //  could be problematic, do not want to change the the rng_states in global mem and get interference between threads

    hiprandState rng = *(ctx->r + id) ; 

    quad4 p ;   
    ctx->scint_photon(p, rng); 

    photon[id] = p ; 
}

extern "C" void QCtx_generate_photon(dim3 numBlocks, dim3 threadsPerBlock, qctx* ctx, quad4* photon, unsigned num_photon ) 
{
    printf("//QCtx_generate_photon num_photon %d \n", num_photon ); 
    _QCtx_generate_photon<<<numBlocks,threadsPerBlock>>>( ctx, photon, num_photon );
} 




__global__ void _QCtx_boundary_lookup_all(qctx* ctx, quad* lookup, unsigned width, unsigned height )
{
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned index = iy * width + ix ;
    if (ix >= width | iy >= height ) return;

    quad q ; 
    q.f = ctx->boundary_lookup( ix, iy ); 
    lookup[index] = q ; 
}

extern "C" void QCtx_boundary_lookup_all(dim3 numBlocks, dim3 threadsPerBlock, qctx* ctx, quad* lookup, unsigned width, unsigned height )
{
    printf("//QCtx_boundary_lookup width %d  height %d \n", width, height ); 
    _QCtx_boundary_lookup_all<<<numBlocks,threadsPerBlock>>>( ctx, lookup, width, height );
}



__global__ void _QCtx_boundary_lookup_line(qctx* ctx, quad* lookup, float* domain, unsigned num_lookup, unsigned line, unsigned k )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= num_lookup) return;
    float wavelength = domain[id] ;  
    quad q ; 
    q.f = ctx->boundary_lookup( wavelength, line, k ); 
    lookup[id] = q ; 
}


extern "C" void QCtx_boundary_lookup_line(dim3 numBlocks, dim3 threadsPerBlock, qctx* ctx, quad* lookup, float* domain, unsigned num_lookup, unsigned line, unsigned k )
{
    printf("//QCtx_boundary_lookup_line num_lookup %d line %d k %d  \n", num_lookup, line, k ); 
    _QCtx_boundary_lookup_line<<<numBlocks,threadsPerBlock>>>( ctx, lookup, domain, num_lookup, line, k );
}



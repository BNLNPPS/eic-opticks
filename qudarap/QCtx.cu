#include "hip/hip_runtime.h"
/**

REMEMBER TO KEEP CODE HERE TO A MINIMUM : PUT AS MUCH AS POSSIBLE INTO THE 
MORE EASILY TESTED FROM MULTIPLE ENVIRONMENTS HEADERS 

**/

#include "stdio.h"
#include "hiprand/hiprand_kernel.h"
#include "scuda.h"
#include "qgs.h"
#include "qctx.h"

/**
HMM hd_factor is more appropriate as a property of the uploaded texture than it is an input argument 
TODO: rearrange hd_factor 
**/

__global__ void _QCtx_generate_scint_wavelength(qctx* ctx, float* wavelength, unsigned num_wavelength, unsigned hd_factor )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= num_wavelength) return;

    hiprandState rng = *(ctx->r + id) ; 

    float wl ; 
    switch(hd_factor)
    {
        case 0:  wl = ctx->scint_wavelength_hd0(rng)  ; break ; 
        case 10: wl = ctx->scint_wavelength_hd10(rng) ; break ; 
        case 20: wl = ctx->scint_wavelength_hd20(rng) ; break ; 
        default: wl = 0.f ; 
    }
    if(id % 100000 == 0) printf("//_QCtx_generate_scint_wavelength id %d hd_factor %d wl %10.4f    \n", id, hd_factor, wl  ); 
    wavelength[id] = wl ; 
}

extern "C" void QCtx_generate_scint_wavelength(dim3 numBlocks, dim3 threadsPerBlock, qctx* ctx, float* wavelength, unsigned num_wavelength, unsigned hd_factor ) 
{
    printf("//QCtx_generate_scint_wavelength num_wavelength %d \n", num_wavelength ); 
    _QCtx_generate_scint_wavelength<<<numBlocks,threadsPerBlock>>>( ctx, wavelength, num_wavelength, hd_factor );
} 

/**
genstep provisioning ? gensteps need to be uploaded with pointer held in qctx 
but for testing need to be able to manually fabricate a genstep
**/

__global__ void _QCtx_generate_cerenkov_wavelength(qctx* ctx, float* wavelength, unsigned num_wavelength )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= num_wavelength) return;

    hiprandState rng = *(ctx->r + id) ; 

    QG qg ;      
    qg.zero();  

    GS& g = qg.g ; 

    // fabricate some values for the genstep
    g.st.Id = 0 ; 
    g.st.ParentId = 0 ; 
    g.st.MaterialIndex = 0 ; 
    g.st.NumPhotons = 0 ; 

    g.st.x0.x = 100.f ; 
    g.st.x0.y = 100.f ; 
    g.st.x0.z = 100.f ; 
    g.st.t0 = 20.f ; 

    g.st.DeltaPosition.x = 1000.f ; 
    g.st.DeltaPosition.y = 1000.f ; 
    g.st.DeltaPosition.z = 1000.f ; 
    g.st.step_length = 1000.f ; 

    g.ck1.code = 0 ; 
    g.ck1.charge = 1.f ; 
    g.ck1.weight = 1.f ; 
    g.ck1.preVelocity = 0.f ; 

    g.ck1.BetaInverse = 1.2f ; 
    g.ck1.Wmin = 300.f ; 
    g.ck1.Wmax = 600.f ; 
    g.ck1.maxCos = 0.f ; 

    g.ck1.maxSin2 = 0.f ; 
    g.ck1.MeanNumberOfPhotons1 = 0.f ; 
    g.ck1.MeanNumberOfPhotons2 = 0.f ; 
    g.ck1.postVelocity = 0.f ; 

    float wl = ctx->cerenkov_wavelength(g, rng);   

    if(id % 100000 == 0) printf("//_QCtx_generate_cerenkov_wavelength id %d wl %10.4f    \n", id, wl  ); 
    wavelength[id] = wl ; 
}


extern "C" void QCtx_generate_cerenkov_wavelength(dim3 numBlocks, dim3 threadsPerBlock, qctx* ctx, float* wavelength, unsigned num_wavelength ) 
{
    printf("//QCtx_generate_cerenkov_wavelength num_wavelength %d \n", num_wavelength ); 
    _QCtx_generate_cerenkov_wavelength<<<numBlocks,threadsPerBlock>>>( ctx, wavelength, num_wavelength );
} 



__global__ void _QCtx_generate_photon(qctx* ctx, quad4* photon, unsigned num_photon )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= num_photon) return;
    
    //ctx->r += id ;   
    //  could be problematic, do not want to change the the rng_states in global mem and get interference between threads

    hiprandState rng = *(ctx->r + id) ; 

    // TODO: this kinda stuff, all non-glue stuff,  should be in headers, not here 

    QG qg ;      
    qg.zero();  

    GS& g = qg.g ; 

    // fabricate some values for the genstep
    g.st.Id = 0 ; 
    g.st.ParentId = 0 ; 
    g.st.MaterialIndex = 0 ; 
    g.st.NumPhotons = 0 ; 

    g.st.x0.x = 100.f ; 
    g.st.x0.y = 100.f ; 
    g.st.x0.z = 100.f ; 
    g.st.t0 = 20.f ; 

    g.st.DeltaPosition.x = 1000.f ; 
    g.st.DeltaPosition.y = 1000.f ; 
    g.st.DeltaPosition.z = 1000.f ; 
    g.st.step_length = 1000.f ; 

    g.sc1.code = 1 ; 
    g.sc1.charge = 1.f ;
    g.sc1.weight = 1.f ;
    g.sc1.midVelocity = 0.f ; 

    g.sc1.scnt = 0 ;
    g.sc1.f41 = 0.f ;   
    g.sc1.f42 = 0.f ;   
    g.sc1.f43 = 0.f ;   

    g.sc1.ScintillationTime = 10.f ;
    g.sc1.f51 = 0.f ;
    g.sc1.f52 = 0.f ;
    g.sc1.f53 = 0.f ;


    quad4 p ;   
    ctx->scint_photon(p, g, rng); 

    photon[id] = p ; 
}

extern "C" void QCtx_generate_photon(dim3 numBlocks, dim3 threadsPerBlock, qctx* ctx, quad4* photon, unsigned num_photon ) 
{
    printf("//QCtx_generate_photon num_photon %d \n", num_photon ); 
    _QCtx_generate_photon<<<numBlocks,threadsPerBlock>>>( ctx, photon, num_photon );
} 




__global__ void _QCtx_boundary_lookup_all(qctx* ctx, quad* lookup, unsigned width, unsigned height )
{
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned index = iy * width + ix ;
    if (ix >= width | iy >= height ) return;

    quad q ; 
    q.f = ctx->boundary_lookup( ix, iy ); 
    lookup[index] = q ; 
}

extern "C" void QCtx_boundary_lookup_all(dim3 numBlocks, dim3 threadsPerBlock, qctx* ctx, quad* lookup, unsigned width, unsigned height )
{
    printf("//QCtx_boundary_lookup width %d  height %d \n", width, height ); 
    _QCtx_boundary_lookup_all<<<numBlocks,threadsPerBlock>>>( ctx, lookup, width, height );
}



__global__ void _QCtx_boundary_lookup_line(qctx* ctx, quad* lookup, float* domain, unsigned num_lookup, unsigned line, unsigned k )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= num_lookup) return;
    float wavelength = domain[id] ;  
    quad q ; 
    q.f = ctx->boundary_lookup( wavelength, line, k ); 
    lookup[id] = q ; 
}


extern "C" void QCtx_boundary_lookup_line(dim3 numBlocks, dim3 threadsPerBlock, qctx* ctx, quad* lookup, float* domain, unsigned num_lookup, unsigned line, unsigned k )
{
    printf("//QCtx_boundary_lookup_line num_lookup %d line %d k %d  \n", num_lookup, line, k ); 
    _QCtx_boundary_lookup_line<<<numBlocks,threadsPerBlock>>>( ctx, lookup, domain, num_lookup, line, k );
}



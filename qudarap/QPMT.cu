#include "hip/hip_runtime.h"
/**
QPMT.cu
==========

_QPMT_lpmtcat_rindex
_QPMT_lpmtcat_qeshape
_QPMT_lpmtcat_stackspec
    kernel funcs taking (qpmt,lookup,domain,domain_width) args

QPMT_lpmtcat
    CPU entry point to launch above kernels controlled by etype


_QPMT_lpmtid_stackspec
    kernel funcs taking (qpmt,lookup,domain,domain_width,lpmtid,num_lpmtid) args

_QPMT_mct_lpmtid
    payload size P templated kernel function with domain and lpmtid array inputs

    * within lpmtid loop calls qpmt.h method depending on etype
    * etype : (qpmt_SPEC qpmt_LL qpmt_COMP qpmt_ART qpmt_ARTE)

QPMT_mct_lpmtid
    CPU entry point to launch above kernel passing etype


**/

#include "QUDARAP_API_EXPORT.hh"
#include <stdio.h>
#include "qpmt_enum.h"
#include "qpmt.h"
#include "qprop.h"


/**
_QPMT_lpmtcat_rindex
---------------------------

max_iprop::

   . (ni-1)*nj*nk + (nj-1)*nk + (nk-1)
   =  ni*nj*nk - nj*nk + nj*nk - nk + nk - 1
   =  ni*nj*nk - 1


HMM: not so easy to generalize from rindex to also do qeshape
because of the different array shapes

Each thread does all pmtcat,layers and props for a single energy_eV.

**/

template <typename F>
__global__ void _QPMT_lpmtcat_rindex( qpmt<F>* pmt, F* lookup , const F* domain, unsigned domain_width )
{
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix >= domain_width ) return;
    F energy_eV = domain[ix] ;

    //printf("//_QPMT_rindex domain_width %d ix %d energy_eV %10.4f \n", domain_width, ix, energy_eV );
    // wierd unsigned/int diff between qpmt.h and here ? to get it to compile fo device
    // switching to enum rather than constexpr const avoids the wierdness

    const int& ni = qpmt_NUM_CAT ;
    const int& nj = qpmt_NUM_LAYR ;
    const int& nk = qpmt_NUM_PROP ;

    //printf("//_QPMT_lpmtcat_rindex ni %d nj %d nk %d \n", ni, nj, nk );
    // cf the CPU equivalent NP::combined_interp_5

    for(int i=0 ; i < ni ; i++)
    for(int j=0 ; j < nj ; j++)
    for(int k=0 ; k < nk ; k++)
    {
        int iprop = i*nj*nk+j*nk+k ;            // linearized higher dimensions
        int index = iprop * domain_width + ix ; // output index into lookup

        F value = pmt->rindex_prop->interpolate(iprop, energy_eV );

        //printf("//_QPMT_lpmtcat_rindex iprop %d index %d value %10.4f \n", iprop, index, value );

        lookup[index] = value ;
    }
}


template <typename F>
__global__ void _QPMT_lpmtcat_qeshape( qpmt<F>* pmt, F* lookup , const F* domain, unsigned domain_width )
{
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix >= domain_width ) return;
    F energy_eV = domain[ix] ;

    //printf("//_QPMT_lpmtcat_qeshape domain_width %d ix %d energy_eV %10.4f \n", domain_width, ix, energy_eV );

    const int& ni = qpmt_NUM_CAT ;

    for(int i=0 ; i < ni ; i++)
    {
        F value = pmt->qeshape_prop->interpolate(i, energy_eV );

        int index = i * domain_width + ix ; // output index into lookup
        lookup[index] = value ;
    }
}



template <typename F>
__global__ void _QPMT_lpmtcat_cetheta( qpmt<F>* pmt, F* lookup , const F* domain, unsigned domain_width )
{
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix >= domain_width ) return;
    F theta_radians = domain[ix] ;

    //printf("//_QPMT_lpmtcat_cetheta domain_width %d ix %d theta_radians %10.4f \n", domain_width, ix, theta_radians );

    const int& ni = qpmt_NUM_CAT ;

    for(int i=0 ; i < ni ; i++)
    {
        F value = pmt->cetheta_prop->interpolate(i, theta_radians );

        int index = i * domain_width + ix ; // output index into lookup
        lookup[index] = value ;
    }
}





template <typename F>
__global__ void _QPMT_lpmtcat_stackspec( qpmt<F>* pmt, F* lookup , const F* domain, unsigned domain_width )
{
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix >= domain_width ) return;
    F energy_eV = domain[ix] ;

    //printf("//_QPMT_lpmtcat_stackspec domain_width %d ix %d energy_eV %10.4f \n", domain_width, ix, energy_eV );

    const int& ni = qpmt_NUM_CAT ;
    const int& nj = domain_width ;
    const int  nk = 16 ;
    const int&  j = ix ;

    F ss[nk] ;

    for(int i=0 ; i < ni ; i++)  // over pmtcat
    {
        int index = i*nj*nk + j*nk  ;
        pmt->get_lpmtcat_stackspec(ss, i, energy_eV );
        for( int k=0 ; k < nk ; k++) lookup[index+k] = ss[k] ;
    }
}


template <typename F> extern void QPMT_lpmtcat(
    dim3 numBlocks,
    dim3 threadsPerBlock,
    qpmt<F>* pmt,
    int etype,
    F* lookup,
    const F* domain,
    unsigned domain_width
)
{
    switch(etype)
    {
        case qpmt_RINDEX   : _QPMT_lpmtcat_rindex<F><<<numBlocks,threadsPerBlock>>>( pmt, lookup, domain, domain_width )    ; break ;
        case qpmt_QESHAPE  : _QPMT_lpmtcat_qeshape<F><<<numBlocks,threadsPerBlock>>>( pmt, lookup, domain, domain_width )   ; break ;
        case qpmt_CETHETA  : _QPMT_lpmtcat_cetheta<F><<<numBlocks,threadsPerBlock>>>( pmt, lookup, domain, domain_width )   ; break ;
        case qpmt_CATSPEC  : _QPMT_lpmtcat_stackspec<F><<<numBlocks,threadsPerBlock>>>( pmt, lookup, domain, domain_width ) ; break ;
    }
}

template void QPMT_lpmtcat(
   dim3,
   dim3,
   qpmt<float>*,
   int etype,
   float*,
   const float* ,
   unsigned
  );


/**
_QPMT_lpmtid_stackspec
-------------------------

**/


template <typename F>
__global__ void _QPMT_lpmtid_stackspec(
    qpmt<F>* pmt,
    F* lookup ,
    const F* domain,
    unsigned domain_width,
    const int* lpmtid,
    unsigned num_lpmtid )
{
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix >= domain_width ) return;
    F energy_eV = domain[ix] ;

    const int& ni = num_lpmtid ;
    const int& nj = domain_width ;
    const int  nk = 16 ;
    const int&  j = ix ;

    F ss[nk] ;

    for(int i=0 ; i < ni ; i++)  // over num_lpmtid
    {
        int pmtid = lpmtid[i] ;
        int index = i*nj*nk + j*nk  ;
        pmt->get_lpmtid_stackspec(ss, pmtid, energy_eV );
        for( int k=0 ; k < nk ; k++) lookup[index+k] = ss[k] ;
    }
}


#ifdef WITH_CUSTOM4
// templated payload size P as it needs to be a compile time constant
template <typename F, int P>
__global__ void _QPMT_mct_lpmtid(
    qpmt<F>* pmt,
    int etype,
    F* lookup ,
    const F* domain,
    unsigned domain_width,
    const int* lpmtid,
    unsigned num_lpmtid )
{
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix >= domain_width ) return;

    //printf("//_QPMT_mct_lpmtid ix %d num_lpmtid %d P %d \n", ix, num_lpmtid, P );

    F minus_cos_theta = domain[ix] ;
    F wavelength_nm = 440.f ;
    F dot_pol_cross_mom_nrm = 0.f ; // SPOL zero is pure P polarized

    const int& ni = num_lpmtid ;
    const int& nj = domain_width ;   // minus_cos_theta values "AOI"
    const int&  j = ix ;

    F payload[P] ;

    for(int i=0 ; i < ni ; i++)  // over num_lpmtid
    {
        int index = i*nj*P + j*P  ;
        int pmtid = lpmtid[i] ;

        if( etype == qpmt_SPEC )
        {
            pmt->get_lpmtid_SPEC(payload, pmtid, wavelength_nm, minus_cos_theta, dot_pol_cross_mom_nrm );
        }
        else if( etype == qpmt_LL )
        {
            pmt->get_lpmtid_LL(payload, pmtid, wavelength_nm, minus_cos_theta, dot_pol_cross_mom_nrm );
        }
        else if( etype == qpmt_COMP )
        {
            pmt->get_lpmtid_COMP(payload, pmtid, wavelength_nm, minus_cos_theta, dot_pol_cross_mom_nrm );
        }
        else if( etype == qpmt_ART )
        {
            pmt->get_lpmtid_ART(payload, pmtid, wavelength_nm, minus_cos_theta, dot_pol_cross_mom_nrm );
        }
        else if( etype == qpmt_ARTE )
        {
            pmt->get_lpmtid_ARTE(payload, pmtid, wavelength_nm, minus_cos_theta, dot_pol_cross_mom_nrm );
        }

        for( int k=0 ; k < P ; k++) lookup[index+k] = payload[k] ;
    }
}


template <typename F> extern void QPMT_mct_lpmtid(
    dim3 numBlocks,
    dim3 threadsPerBlock,
    qpmt<F>* pmt,
    int etype,
    F* lookup,
    const F* domain,
    unsigned domain_width,
    const int* lpmtid,
    unsigned num_lpmtid
)
{
    printf("//QPMT_mct_lpmtid etype %d domain_width %d num_lpmtid %d \n", etype, domain_width, num_lpmtid);

    switch(etype)
    {
        case qpmt_SPEC:
           _QPMT_mct_lpmtid<F,16><<<numBlocks,threadsPerBlock>>>(
              pmt, etype, lookup, domain, domain_width, lpmtid, num_lpmtid ) ;  break ;

        case qpmt_ART:
           _QPMT_mct_lpmtid<F,16><<<numBlocks,threadsPerBlock>>>(
              pmt, etype, lookup, domain, domain_width, lpmtid, num_lpmtid ) ;  break ;

        case qpmt_COMP:
           _QPMT_mct_lpmtid<F,32><<<numBlocks,threadsPerBlock>>>(
              pmt, etype, lookup, domain, domain_width, lpmtid, num_lpmtid ) ;  break ;

        case qpmt_LL:
           _QPMT_mct_lpmtid<F,128><<<numBlocks,threadsPerBlock>>>(
              pmt, etype, lookup, domain, domain_width, lpmtid, num_lpmtid ) ;  break ;

        case qpmt_ARTE:
           _QPMT_mct_lpmtid<F,4><<<numBlocks,threadsPerBlock>>>(
              pmt, etype, lookup, domain, domain_width, lpmtid, num_lpmtid ) ;  break ;
    }
}

template void QPMT_mct_lpmtid<float>(   dim3, dim3, qpmt<float>*, int etype, float*,  const float* , unsigned, const int*, unsigned);
#endif


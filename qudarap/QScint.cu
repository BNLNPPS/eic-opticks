

#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"

__global__ void _QScint_generate_kernel(hiprandState* rng_states, hipTextureObject_t texObj, float* wavelength, unsigned num_wavelength )
{
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= num_wavelength) return;

    float u = hiprand_uniform(&rng_states[id]); 

    float wl = tex2D<float>(texObj,  u, 0.f);

    wavelength[id] = wl ;   
}

extern "C" void QScint_generate_kernel(dim3 numBlocks, dim3 threadsPerBlock, hiprandState* rng_states, hipTextureObject_t texObj, float* wavelength, unsigned num_wavelength ) 
{
    _QScint_generate_kernel<<<numBlocks,threadsPerBlock>>>( rng_states, texObj, wavelength, num_wavelength );
} 



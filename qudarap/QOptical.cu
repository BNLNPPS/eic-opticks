#include "hip/hip_runtime.h"

#include <stdio.h>
#include "scuda.h"
#include "squad.h"


__global__ void _QOptical_check( quad* optical, unsigned width, unsigned height )
{
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned index = iy * width + ix ;
    if (ix >= width | iy >= height ) return;

    uint4& u = optical[iy].u ; 

    printf("//_QOptical_check ix %d iy %d index %d  optical[iy] (%d %d %d %d)   \n", ix, iy, index, u.x, u.y, u.z, u.w ); 
}

extern "C" void QOptical_check(dim3 numBlocks, dim3 threadsPerBlock, quad* optical, unsigned width, unsigned height ) 
{
    _QOptical_check<<<numBlocks,threadsPerBlock>>>( optical, width, height );
} 



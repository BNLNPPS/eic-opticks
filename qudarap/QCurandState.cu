#include "hip/hip_runtime.h"
#include <cstdio>
#include "hiprand/hiprand_kernel.h"
#include "qcurandstate.h"
#include "QUDA_CHECK.h"
#include "SLaunchSequence.h"



__global__ void _QCurandState_curand_init(int threads_per_launch, int thread_offset, qcurandstate* cs, hiprandState* states_thread_offset )
{
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= threads_per_launch) return;
    hiprand_init(cs->seed, id+thread_offset, cs->offset, states_thread_offset + id );  

    //if( id == 0 ) printf("// _QCurandState_curand_init thread_offset %d \n", thread_offset ); 
}


void before_kernel( hipEvent_t& start, hipEvent_t& stop )
{
    QUDA_CHECK( hipEventCreate( &start ) );
    QUDA_CHECK( hipEventCreate( &stop ) );
    QUDA_CHECK( hipEventRecord( start,0 ) );
}
float after_kernel( hipEvent_t& start, hipEvent_t& stop )
{
    float kernel_time = 0.f ;

    QUDA_CHECK( hipEventRecord( stop,0 ) );
    QUDA_CHECK( hipEventSynchronize(stop) );

    QUDA_CHECK( hipEventElapsedTime(&kernel_time, start, stop) );
    QUDA_CHECK( hipEventDestroy( start ) );
    QUDA_CHECK( hipEventDestroy( stop ) );

    QUDA_CHECK( hipDeviceSynchronize() );

    return kernel_time ;
}

extern "C" void QCurandState_curand_init(SLaunchSequence* seq,  qcurandstate* cs, qcurandstate* d_cs) 
{
    // NB this is still on CPU, dereferencing d_cs here will BUS_ERROR 

    printf("//QCurandState_curand_init seq.items %d cs %p  d_cs %p cs.num %llu \n", seq->items, cs, d_cs, cs->num );  

    hipEvent_t start, stop ;

    for(unsigned i=0 ; i < seq->launches.size() ; i++)
    {
        SLaunch& l = seq->launches[i] ; 
        printf("// l.sequence_index %d  l.blocks_per_launch %d l.threads_per_block %d  l.threads_per_launch %d l.thread_offset %d  \n", 
                   l.sequence_index,    l.blocks_per_launch,   l.threads_per_block,    l.threads_per_launch,   l.thread_offset  );  

        hiprandState* states_thread_offset = cs->states  + l.thread_offset ; 
     
        before_kernel( start, stop );

        _QCurandState_curand_init<<<l.blocks_per_launch,l.threads_per_block>>>( l.threads_per_launch, l.thread_offset, d_cs, states_thread_offset  );  

        l.kernel_time = after_kernel( start, stop ); 
    }

} 





#include "hip/hip_runtime.h"
#include <cstdio>
#include "hiprand/hiprand_kernel.h"
#include "qcurandstate.h"
#include "QUDA_CHECK.h"
#include "SLaunchSequence.h"

#include "scurandref.h"


__global__ void _QCurandState_curand_init(int threads_per_launch, int id_offset, qcurandstate* cs, hiprandState* states_thread_offset )
{
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= threads_per_launch) return;
    hiprand_init(cs->seed, id+id_offset, cs->offset, states_thread_offset + id );  

    //if( id == 0 ) printf("// _QCurandState_curand_init id_offset %d \n", id_offset ); 
}


/**
_QCurandState_curand_init_chunk
---------------------------------

id 
   [0:threads_per_launch]

states_thread_offset 
   enables multiple launches to write into the correct output slot

**/


__global__ void _QCurandState_curand_init_chunk(int threads_per_launch, int id_offset, scurandref* cr, hiprandState* states_thread_offset )
{
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= threads_per_launch) return;
    hiprand_init(cr->seed, id+id_offset, cr->offset, states_thread_offset + id );  

    //if( id == 0 ) printf("// _QCurandState_curand_init_chunk id_offset %d \n", id_offset ); 
}








void before_kernel( hipEvent_t& start, hipEvent_t& stop )
{
    QUDA_CHECK( hipEventCreate( &start ) );
    QUDA_CHECK( hipEventCreate( &stop ) );
    QUDA_CHECK( hipEventRecord( start,0 ) );
}
float after_kernel( hipEvent_t& start, hipEvent_t& stop )
{
    float kernel_time = 0.f ;

    QUDA_CHECK( hipEventRecord( stop,0 ) );
    QUDA_CHECK( hipEventSynchronize(stop) );

    QUDA_CHECK( hipEventElapsedTime(&kernel_time, start, stop) );
    QUDA_CHECK( hipEventDestroy( start ) );
    QUDA_CHECK( hipEventDestroy( stop ) );

    QUDA_CHECK( hipDeviceSynchronize() );

    return kernel_time ;
}


extern "C" void QCurandState_curand_init(SLaunchSequence* seq,  qcurandstate* cs, qcurandstate* d_cs) 
{
    // NB this is still on CPU, dereferencing d_cs here will BUS_ERROR 

    printf("//QCurandState_curand_init seq.items %d cs %p  d_cs %p cs.num %llu \n", seq->items, cs, d_cs, cs->num );  

    hipEvent_t start, stop ;

    for(unsigned i=0 ; i < seq->launches.size() ; i++)
    {
        SLaunch& l = seq->launches[i] ; 
        printf("// l.sequence_index %d  l.blocks_per_launch %d l.threads_per_block %d  l.threads_per_launch %d l.thread_offset %d  \n", 
                   l.sequence_index,    l.blocks_per_launch,   l.threads_per_block,    l.threads_per_launch,   l.thread_offset  );  

        int id_offset = l.thread_offset ;   

        hiprandState* states_thread_offset = cs->states  + l.thread_offset ; 
     
        before_kernel( start, stop );

        _QCurandState_curand_init<<<l.blocks_per_launch,l.threads_per_block>>>( l.threads_per_launch, id_offset, d_cs, states_thread_offset  );  

        l.kernel_time = after_kernel( start, stop ); 
    }

} 


/**
QCurandState_curand_init_chunk
--------------------------------

NB cr and d_cr hold the same values, however cr is host pointer and d_cr is device pointer
cr->states is device pointer, note that pointer arithmetic works on device pointer 

Because are writing states just for a chunk do not need a chunk_offset on the output side,
but do need chunk_offset for the input side. 

**/

extern "C" void QCurandState_curand_init_chunk(SLaunchSequence* seq,  scurandref* cr, scurandref* d_cr) 
{
    // NB this is still on CPU, dereferencing d_cs here will BUS_ERROR 

    printf("//QCurandState_curand_init_chunk seq.items %d cr %p  d_cr %p cr.num %llu cr.chunk_offset %llu \n", seq->items, cr, d_cr, cr->num, cr->chunk_offset );  

    hipEvent_t start, stop ;

    for(unsigned i=0 ; i < seq->launches.size() ; i++)
    {
        SLaunch& l = seq->launches[i] ; 

        if(0) printf("// l.sequence_index %d  l.blocks_per_launch %d l.threads_per_block %d  l.threads_per_launch %d l.thread_offset %d  \n", 
                         l.sequence_index,    l.blocks_per_launch,   l.threads_per_block,    l.threads_per_launch,   l.thread_offset  );  

        int id_offset = l.thread_offset + cr->chunk_offset ;   

        hiprandState* states_thread_offset = cr->states  + l.thread_offset ; 
     
        before_kernel( start, stop );

        _QCurandState_curand_init_chunk<<<l.blocks_per_launch,l.threads_per_block>>>( l.threads_per_launch, id_offset, d_cr, states_thread_offset  );  

        l.kernel_time = after_kernel( start, stop ); 
    }

} 




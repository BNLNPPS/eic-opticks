#include "hip/hip_runtime.h"
/**
QSim.cu : extern void CUDA launch functions testing qsim.h methods
-------------------------------------------------------------------------------------

The launch functions are all invoked from QSim.cc methods with corresponding names.   




**/



#include "stdio.h"
#include "hiprand/hiprand_kernel.h"
#include "scuda.h"
#include "qgs.h"
#include "qprop.h"
#include "qsim.h"
#include "qcurand.h"
#include "qevent.h"
#include "qdebug.h"

#include "QSimLaunch.hh"


/**
_QSim_rng_sequence
--------------------

id_offset : applies to sim.rngstate array controlling which hiprandState to use

**/

template <typename T>
__global__ void _QSim_rng_sequence(qsim<T>* sim, T* seq, unsigned ni, unsigned nv, unsigned id_offset )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= ni) return;
    hiprandState rng = sim->rngstate[id+id_offset]; 
    unsigned ibase = id*nv ; 

    for(unsigned v=0 ; v < nv ; v++)
    {
        T u = qcurand<T>::uniform(&rng) ;
        seq[ibase+v] = u ;
    } 
}


template <typename T>
extern void QSim_rng_sequence(dim3 numBlocks, dim3 threadsPerBlock, qsim<T>* sim, T*  seq, unsigned ni, unsigned nv, unsigned id_offset )
{
    printf("//QSim_rng_sequence_f ni %d nv %d id_offset %d  \n", ni, nv, id_offset ); 
    _QSim_rng_sequence<T><<<numBlocks,threadsPerBlock>>>( sim, seq, ni, nv, id_offset );

}

template void QSim_rng_sequence(dim3, dim3, qsim<float>*, float*, unsigned, unsigned, unsigned); 
template void QSim_rng_sequence(dim3, dim3, qsim<double>*, double*, unsigned, unsigned, unsigned); 






/**
HMM hd_factor is more appropriate as a property of the uploaded texture than it is an input argument 
TODO: rearrange hd_factor 
**/

template<typename T>
__global__ void _QSim_scint_wavelength(qsim<T>* sim, T* wavelength, unsigned num_wavelength, unsigned hd_factor )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= num_wavelength) return;

    hiprandState rng = sim->rngstate[id]; 

    T wl ; 
    switch(hd_factor)
    {
        case 0:  wl = sim->scint_wavelength_hd0(rng)  ; break ; 
        case 10: wl = sim->scint_wavelength_hd10(rng) ; break ; 
        case 20: wl = sim->scint_wavelength_hd20(rng) ; break ; 
        default: wl = 0.f ; 
    }
    if(id % 100000 == 0) printf("//_QSim_scint_wavelength id %d hd_factor %d wl %10.4f    \n", id, hd_factor, wl  ); 
    wavelength[id] = wl ; 
}

template <typename T>
extern void QSim_scint_wavelength(dim3 numBlocks, dim3 threadsPerBlock, qsim<T>* sim, T* wavelength, unsigned num_wavelength, unsigned hd_factor ) 
{
    printf("//QSim_scint_wavelength num_wavelength %d \n", num_wavelength ); 
    _QSim_scint_wavelength<T><<<numBlocks,threadsPerBlock>>>( sim, wavelength, num_wavelength, hd_factor );
} 


template void QSim_scint_wavelength(dim3, dim3, qsim<double>*, double*, unsigned, unsigned ); 
template void QSim_scint_wavelength(dim3, dim3, qsim<float>*, float*, unsigned, unsigned ); 






/**
genstep provisioning ? gensteps need to be uploaded with pointer held in qsim 
but for testing need to be able to manually fabricate a genstep
**/

template <typename T>
__global__ void _QSim_cerenkov_wavelength_rejection_sampled(qsim<T>* sim, T* wavelength, unsigned num_wavelength )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= num_wavelength) return;

    hiprandState rng = sim->rngstate[id]; 

    T wl = sim->cerenkov_wavelength_rejection_sampled(id, rng);   

    if(id % 100000 == 0) printf("//_QSim_cerenkov_wavelength_rejection_sampled id %d wl %10.4f    \n", id, wl  ); 
    wavelength[id] = wl ; 
}


template <typename T>
extern void QSim_cerenkov_wavelength_rejection_sampled(dim3 numBlocks, dim3 threadsPerBlock, qsim<T>* sim, T* wavelength, unsigned num_wavelength ) 
{
    printf("//QSim_cerenkov_wavelength_rejection_sampled num_wavelength %d \n", num_wavelength ); 
    _QSim_cerenkov_wavelength_rejection_sampled<T><<<numBlocks,threadsPerBlock>>>( sim, wavelength, num_wavelength );
} 

template void QSim_cerenkov_wavelength_rejection_sampled(dim3, dim3, qsim<double>*, double*, unsigned ); 
template void QSim_cerenkov_wavelength_rejection_sampled(dim3, dim3, qsim<float>*, float*, unsigned ); 




template <typename T>
__global__ void _QSim_cerenkov_photon(qsim<T>* sim, quad4* photon, unsigned num_photon, int print_id )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= num_photon) return;

    hiprandState rng = sim->rngstate[id]; 

    quad4 p ;   
    sim->cerenkov_photon(p, id, rng, print_id);   

    if(id % 100000 == 0) printf("//_QSim_cerenkov_photon id %d \n", id  ); 
    photon[id] = p ; 
}

template <typename T>
extern void QSim_cerenkov_photon(dim3 numBlocks, dim3 threadsPerBlock, qsim<T>* sim, quad4* photon, unsigned num_photon, int print_id ) 
{
    printf("//QSim_cerenkov_photon num_photon %d \n", num_photon ); 
    _QSim_cerenkov_photon<T><<<numBlocks,threadsPerBlock>>>( sim, photon, num_photon, print_id );
} 

template void QSim_cerenkov_photon(dim3, dim3, qsim<double>*, quad4*, unsigned, int ); 
template void QSim_cerenkov_photon(dim3, dim3, qsim<float>*, quad4*, unsigned, int ); 








template <typename T>
__global__ void _QSim_cerenkov_photon_enprop(qsim<T>* sim, quad4* photon, unsigned num_photon, int print_id )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= num_photon) return;

    hiprandState rng = sim->rngstate[id]; 

    quad4 p ;   
    sim->cerenkov_photon_enprop(p, id, rng, print_id);   

    if(id % 100000 == 0) printf("//_QSim_cerenkov_photon_enprop id %d \n", id  ); 
    photon[id] = p ; 
}

template <typename T>
extern void QSim_cerenkov_photon_enprop(dim3 numBlocks, dim3 threadsPerBlock, qsim<T>* sim, quad4* photon, unsigned num_photon, int print_id ) 
{
    printf("//QSim_cerenkov_photon_enprop num_photon %d \n", num_photon ); 
    _QSim_cerenkov_photon_enprop<T><<<numBlocks,threadsPerBlock>>>( sim, photon, num_photon, print_id );
} 

template void QSim_cerenkov_photon_enprop(dim3, dim3, qsim<double>*, quad4*, unsigned, int ); 
template void QSim_cerenkov_photon_enprop(dim3, dim3, qsim<float>*, quad4*, unsigned, int ); 








template <typename T>
__global__ void _QSim_cerenkov_photon_expt(qsim<T>* sim, quad4* photon, unsigned num_photon, int print_id )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= num_photon) return;

    hiprandState rng = sim->rngstate[id]; 

    quad4 p ;   
    sim->cerenkov_photon_expt(p, id, rng, print_id);   

    if(id % 100000 == 0) printf("//_QSim_cerenkov_photon_expt id %d \n", id  ); 
    photon[id] = p ; 
}

template <typename T>
extern void QSim_cerenkov_photon_expt(dim3 numBlocks, dim3 threadsPerBlock, qsim<T>* sim, quad4* photon, unsigned num_photon, int print_id ) 
{
    printf("//QSim_cerenkov_photon_expt num_photon %d \n", num_photon ); 
    _QSim_cerenkov_photon_expt<T><<<numBlocks,threadsPerBlock>>>( sim, photon, num_photon, print_id );
} 

template void QSim_cerenkov_photon_expt(dim3, dim3, qsim<double>*, quad4*, unsigned, int ); 
template void QSim_cerenkov_photon_expt(dim3, dim3, qsim<float>*, quad4*, unsigned, int ); 













template <typename T>
__global__ void _QSim_scint_photon(qsim<T>* sim, quad4* photon, unsigned num_photon )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= num_photon) return;
    
    //sim->r += id ;   
    //  would be problematic, do not want to change the the rng_states in global mem and get interference between threads

    hiprandState rng = sim->rngstate[id] ; 

    quad4 p ;   
    sim->scint_photon(p, rng); 

    photon[id] = p ; 
}

template <typename T>
extern void QSim_scint_photon(dim3 numBlocks, dim3 threadsPerBlock, qsim<T>* sim, quad4* photon, unsigned num_photon ) 
{
    printf("//QSim_scint_photon num_photon %d \n", num_photon ); 
    _QSim_scint_photon<T><<<numBlocks,threadsPerBlock>>>( sim, photon, num_photon );
} 

template void QSim_scint_photon(dim3, dim3, qsim<double>*, quad4*, unsigned ); 
template void QSim_scint_photon(dim3, dim3, qsim<float>*, quad4*, unsigned ); 



template <typename T>
__global__ void _QSim_generate_photon(qsim<T>* sim, qevent* evt )
{
    unsigned photon_id = blockIdx.x*blockDim.x + threadIdx.x;
    
   if (photon_id >= evt->num_photon) return;
    
    hiprandState rng = sim->rngstate[photon_id] ; 
    unsigned genstep_id = evt->seed[photon_id] ; 
    const quad6& gs     = evt->genstep[genstep_id] ; 

    printf("//_QSim_generate_photon photon_id %4d evt->num_photon %4d genstep_id %4d  \n", photon_id, evt->num_photon, genstep_id );  

    quad4 p ;   
    sim->generate_photon(p, rng, gs, photon_id, genstep_id ); 

    //p.q0.f.x = 1.f ; p.q0.f.y = 2.f ; p.q0.f.z = 3.f ; 

    evt->photon[photon_id] = p ; 

}

template <typename T>
extern void QSim_generate_photon(dim3 numBlocks, dim3 threadsPerBlock, qsim<T>* sim, qevent* evt ) 
{
    printf("//QSim_generate_photon sim %p evt %p \n", sim, evt ); 
    // NB trying to use the the sim and evt pointers here gives "Bus error" 
    // thats because this is not yet on GPU, despite being compiled by nvcc
    _QSim_generate_photon<T><<<numBlocks,threadsPerBlock>>>( sim, evt );
} 

template void QSim_generate_photon(dim3, dim3, qsim<double>*, qevent* ); 
template void QSim_generate_photon(dim3, dim3, qsim<float>*,  qevent* ); 



template <typename T>
__global__ void _QSim_fill_state_0(qsim<T>* sim, quad6* state,  unsigned num_state, qdebug* dbg )
{
    unsigned state_id = blockIdx.x*blockDim.x + threadIdx.x;
    printf("//_QSim_fill_state_0 state_id %d \n", state_id ); 

    if (state_id >= num_state) return;

    qstate s ; 

    float wavelength = dbg->wavelength ; 
    float cosTheta = dbg->cosTheta ;  
    int boundary = state_id + 1 ; 

    printf("//_QSim_fill_state_0 state_id %d  boundary %d wavelength %10.4f cosTheta %10.4f   \n", state_id, boundary, wavelength, cosTheta );  

    sim->fill_state(s, boundary, wavelength, cosTheta ); 

    state[state_id].q0.f = s.material1 ; 
    state[state_id].q1.f = s.m1group2 ; 
    state[state_id].q2.f = s.material2 ; 
    state[state_id].q3.f = s.surface ; 
    state[state_id].q4.u = s.optical ; 
    state[state_id].q5.u = s.index ; 

    //printf("//_QSim_fill_state_0 s.material1 %10.4f %10.4f %10.4f %10.4f \n", s.material1.x, s.material1.y, s.material1.z, s.material1.w ); 
}

template <typename T>
extern void QSim_fill_state_0(dim3 numBlocks, dim3 threadsPerBlock, qsim<T>* sim, quad6* state, unsigned num_state, qdebug* dbg )
{
    printf("//QSim_fill_state_0 sim %p state %p num_state %d dbg %p \n", sim, state, num_state, dbg ); 
    _QSim_fill_state_0<T><<<numBlocks,threadsPerBlock>>>( sim, state, num_state, dbg  );
} 

template void QSim_fill_state_0(dim3, dim3, qsim<double>*, quad6* , unsigned, qdebug* ); 
template void QSim_fill_state_0(dim3, dim3, qsim<float>* , quad6* , unsigned, qdebug* ); 



template <typename T>
__global__ void _QSim_fill_state_1( qsim<T>* sim, qstate* state,  unsigned num_state, qdebug* dbg )
{
    unsigned state_id = blockIdx.x*blockDim.x + threadIdx.x;
    printf("//_QSim_fill_state_1 blockIdx.x %d blockDim.x %d threadIdx.x %d state_id %d num_state %d \n", blockIdx.x, blockDim.x, threadIdx.x, state_id, num_state ); 

    if (state_id >= num_state) return;


    const float& wavelength = dbg->wavelength ; 
    const float& cosTheta = dbg->cosTheta ;  
    int boundary = state_id + 1 ; // boundary is 1-based

    printf("//_QSim_fill_state_1 state_id %d  boundary %d wavelength %10.4f cosTheta %10.4f   \n", state_id, boundary, wavelength, cosTheta );  

    qstate s ; 
    sim->fill_state(s, boundary, wavelength, cosTheta ); 

    state[state_id] = s ; 

    //printf("//_QSim_fill_state_1 s.material1 %10.4f %10.4f %10.4f %10.4f \n", s.material1.x, s.material1.y, s.material1.z, s.material1.w ); 
}

template <typename T>
extern void QSim_fill_state_1(dim3 numBlocks, dim3 threadsPerBlock, qsim<T>* sim, qstate* state, unsigned num_state, qdebug* dbg )
{
    printf("//QSim_fill_state_1 sim %p state %p num_state %d dbg %p \n", sim, state, num_state, dbg ); 
    _QSim_fill_state_1<T><<<numBlocks,threadsPerBlock>>>( sim, state, num_state, dbg  );
} 

template void QSim_fill_state_1(dim3, dim3, qsim<double>*, qstate* , unsigned, qdebug* ); 
template void QSim_fill_state_1(dim3, dim3, qsim<float>* , qstate* , unsigned, qdebug* ); 



template <typename T>
__global__ void _QSim_rayleigh_scatter_align( qsim<T>* sim, quad4* photon,  unsigned num_photon, qdebug* dbg )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    //printf("//_QSim_rayleigh_scatter_align blockIdx.x %d blockDim.x %d threadIdx.x %d id %d num_photon %d \n", blockIdx.x, blockDim.x, threadIdx.x, id, num_photon ); 

    if (id >= num_photon) return;

    quad4 p = dbg->p ;    // need local copy of photon otherwise would have write interference between threads
    hiprandState rng = sim->rngstate[id] ; 

    sim->rayleigh_scatter_align(p, rng);  

    photon[id] = p ; 
}

template <typename T>
__global__ void _QSim_propagate_to_boundary( qsim<T>* sim, quad4* photon, unsigned num_photon, qdebug* dbg )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    //printf("//_QSim_propagate_to_boundary blockIdx.x %d blockDim.x %d threadIdx.x %d propagate_id %d \n", blockIdx.x, blockDim.x, threadIdx.x, propagate_id ); 

    if (id >= num_photon) return;

    const qprd& prd = dbg->prd ;  // no need for local copy when readonly   
    const qstate& s = dbg->s ;     
    quad4 p         = dbg->p ;    // need local copy of photon otherwise will have write interference between threads

    hiprandState rng = sim->rngstate[id] ; 

    unsigned flag = 0u ;  
    sim->propagate_to_boundary( flag, p, prd, s, rng );  
    p.q3.u.w = flag ;  // non-standard
    photon[id] = p ; 

    const float3* position = (float3*)&p.q0.f.x ; 
    const float& time = p.q0.f.w ; 
    printf("//_QSim_propagate_to_boundary flag %d position %10.4f %10.4f %10.4f  time %10.4f  \n", flag, position->x, position->y, position->z, time ); 

}

template <typename T>
__global__ void _QSim_propagate_at_boundary_generate( qsim<T>* sim, quad4* photon, unsigned num_photon, qdebug* dbg )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    //printf("//_QSim_propagate_at_boundary_generate blockIdx.x %d blockDim.x %d threadIdx.x %d propagate_id %d \n", blockIdx.x, blockDim.x, threadIdx.x, propagate_id ); 

    if (id >= num_photon) return;

    const qprd& prd = dbg->prd ;  // no need for local copy when readonly   
    const qstate& s = dbg->s ;     
    quad4 p         = dbg->p ;    // need local copy of photon otherwise will have write interference between threads
    hiprandState rng = sim->rngstate[id] ; 

    p.q0.f = p.q1.f ;   // non-standard record initial mom and pol into q0, q3
    p.q3.f = p.q2.f ; 
    unsigned flag = sim->propagate_at_boundary( p, prd, s, rng, id );  
    p.q3.u.w = flag ;  // non-standard

    photon[id] = p ; 
}


template <typename T>
__global__ void _QSim_propagate_at_boundary_mutate( qsim<T>* sim, quad4* photon, unsigned num_photon, qdebug* dbg )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    //printf("//_QSim_propagate_at_boundary_mutate blockIdx.x %d blockDim.x %d threadIdx.x %d id %d \n", blockIdx.x, blockDim.x, threadIdx.x, id ); 

    if (id >= num_photon) return;

    const qprd& prd = dbg->prd ; 
    const qstate& s = dbg->s ;     
    quad4 p         = photon[id] ; 
    hiprandState rng = sim->rngstate[id] ; 

    p.q0.f = p.q1.f ;   // non-standard record initial mom and pol into q0, q3
    p.q3.f = p.q2.f ; 
    unsigned flag = sim->propagate_at_boundary( p, prd, s, rng, id );  
    p.q3.u.w = flag ;  // non-standard

    photon[id] = p ; 
}



template <typename T>
__global__ void _QSim_hemisphere_polarized( qsim<T>* sim, quad4* photon, unsigned num_photon, qdebug* dbg, unsigned polz )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= num_photon) return;

    hiprandState rng = sim->rngstate[id] ; 
    const qprd& prd = dbg->prd ;  
    quad4 p         = dbg->p ;   
    bool inwards = true ; 

    sim->hemisphere_polarized( p, polz, inwards,  prd, rng );  

    photon[id] = p ; 
}


template <typename T>
extern void QSim_photon_launch(dim3 numBlocks, dim3 threadsPerBlock, qsim<T>* sim, quad4* photon, unsigned num_photon, qdebug* dbg, unsigned type  )
{
    const char* name = QSimLaunch::Name(type) ; 
    printf("//QSim_photon_launch sim %p photon %p num_photon %d dbg %p type %d name %s \n", sim, photon, num_photon, dbg, type, name ); 
    switch(type)
    {
        case PROPAGATE_TO_BOUNDARY:  _QSim_propagate_to_boundary<T><<<numBlocks,threadsPerBlock>>>(  sim, photon, num_photon, dbg  )   ; break ;

        case RAYLEIGH_SCATTER_ALIGN: _QSim_rayleigh_scatter_align<T><<<numBlocks,threadsPerBlock>>>( sim, photon, num_photon, dbg  )   ; break ;

        case HEMISPHERE_S_POLARIZED: _QSim_hemisphere_polarized<T><<<numBlocks,threadsPerBlock>>>(   sim, photon, num_photon, dbg, 0u  ) ; break ; 
        case HEMISPHERE_P_POLARIZED: _QSim_hemisphere_polarized<T><<<numBlocks,threadsPerBlock>>>(   sim, photon, num_photon, dbg, 1u  ) ; break ; 
        case HEMISPHERE_X_POLARIZED: _QSim_hemisphere_polarized<T><<<numBlocks,threadsPerBlock>>>(   sim, photon, num_photon, dbg, 2u  ) ; break ; 

        case PROPAGATE_AT_BOUNDARY:        
        case PROPAGATE_AT_BOUNDARY_NORMAL_INCIDENCE:        
                             _QSim_propagate_at_boundary_generate<T><<<numBlocks,threadsPerBlock>>>(  sim, photon, num_photon, dbg  )   ; break ;


        case PROPAGATE_AT_BOUNDARY_S_POLARIZED: 
        case PROPAGATE_AT_BOUNDARY_P_POLARIZED:
        case PROPAGATE_AT_BOUNDARY_X_POLARIZED:  
                             _QSim_propagate_at_boundary_mutate<T><<<numBlocks,threadsPerBlock>>>(    sim, photon, num_photon, dbg  ) ; break ;
    }
}

template void QSim_photon_launch(dim3, dim3, qsim<double>* , quad4* , unsigned, qdebug*, unsigned  ); 
template void QSim_photon_launch(dim3, dim3, qsim<float>*  , quad4* , unsigned, qdebug*, unsigned  ); 









template <typename T>
__global__ void _QSim_boundary_lookup_all(qsim<T>* sim, quad* lookup, unsigned width, unsigned height )
{
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned index = iy * width + ix ;
    if (ix >= width | iy >= height ) return;

    quad q ; 
    q.f = sim->boundary_lookup( ix, iy ); 
    lookup[index] = q ; 
}

template <typename T>
extern void QSim_boundary_lookup_all(dim3 numBlocks, dim3 threadsPerBlock, qsim<T>* sim, quad* lookup, unsigned width, unsigned height )
{
    printf("//QSim_boundary_lookup width %d  height %d \n", width, height ); 
    _QSim_boundary_lookup_all<T><<<numBlocks,threadsPerBlock>>>( sim, lookup, width, height );
}

template void QSim_boundary_lookup_all(dim3, dim3, qsim<double>*, quad*, unsigned, unsigned ); 
template void QSim_boundary_lookup_all(dim3, dim3, qsim<float>*, quad*, unsigned, unsigned ); 


template <typename T>
__global__ void _QSim_boundary_lookup_line(qsim<T>* sim, quad* lookup, T* domain, unsigned num_lookup, unsigned line, unsigned k )
{
    unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= num_lookup) return;
    T wavelength = domain[id] ;  
    quad q ; 
    q.f = sim->boundary_lookup( wavelength, line, k ); 
    lookup[id] = q ; 
}


template <typename T>
extern void QSim_boundary_lookup_line(dim3 numBlocks, dim3 threadsPerBlock, qsim<T>* sim, quad* lookup, T* domain, unsigned num_lookup, unsigned line, unsigned k )
{
    printf("//QSim_boundary_lookup_line num_lookup %d line %d k %d  \n", num_lookup, line, k ); 
    _QSim_boundary_lookup_line<T><<<numBlocks,threadsPerBlock>>>( sim, lookup, domain, num_lookup, line, k );
}

template void QSim_boundary_lookup_line(dim3, dim3, qsim<double>*, quad*, double*, unsigned, unsigned, unsigned ); 
template void QSim_boundary_lookup_line(dim3, dim3, qsim<float>*, quad*, float*, unsigned, unsigned, unsigned ); 


template <typename T>
__global__ void _QSim_prop_lookup(qsim<T>* sim, T* lookup, const T* domain, unsigned domain_width, unsigned* pids, unsigned num_pids )
{
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (ix >= domain_width || iy >= num_pids  ) return;

    T x = domain[ix] ;  
    unsigned pid = pids[iy] ; 

    T y = sim->prop->interpolate( pid, x ); 
    lookup[iy*domain_width + ix] = y ; 
}

template <typename T>
extern void QSim_prop_lookup( dim3 numBlocks, dim3 threadsPerBlock, qsim<T>* sim, T* lookup, const T* domain, unsigned domain_width, unsigned* pids, unsigned num_pids )
{
    printf("//QSim_prop_lookup domain_width %d num_pids %d  \n", domain_width, num_pids ); 
    _QSim_prop_lookup<T><<<numBlocks,threadsPerBlock>>>( sim, lookup, domain, domain_width, pids, num_pids );
}


template void QSim_prop_lookup(dim3, dim3, qsim<double>*, double*, double const*, unsigned, unsigned*, unsigned) ; 
template void QSim_prop_lookup(dim3, dim3, qsim<float>*,  float*,  float const*, unsigned, unsigned*, unsigned ) ; 







/**
ipid : index of the lookup outputs for that pid, which may differ from index of the pid   
**/

template <typename T>
__global__ void _QSim_prop_lookup_one(qsim<T>* sim, T* lookup, const T* domain, unsigned domain_width, unsigned num_pids, unsigned pid, unsigned ipid )
{
    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix >= domain_width || pid >= num_pids  ) return;

    T x = domain[ix] ;  
    T y = sim->prop->interpolate( pid, x ); 

    lookup[ipid*domain_width + ix] = y ; 
}

template <typename T>
extern  void QSim_prop_lookup_one(dim3 numBlocks, dim3 threadsPerBlock, qsim<T>* sim, T* lookup, const T* domain, unsigned domain_width, unsigned num_pids, unsigned pid, unsigned ipid )
{
    printf("//QSim_prop_lookup_one domain_width %d num_pids %d pid %d ipid %d \n", domain_width, num_pids, pid, ipid ); 
    _QSim_prop_lookup_one<T><<<numBlocks,threadsPerBlock>>>( sim, lookup, domain, domain_width, num_pids, pid, ipid );
}

template void QSim_prop_lookup_one(dim3, dim3, qsim<double>*, double*, const double*, unsigned, unsigned, unsigned, unsigned ) ; 
template void QSim_prop_lookup_one(dim3, dim3, qsim<float>*, float*, const float*, unsigned, unsigned, unsigned, unsigned ) ; 





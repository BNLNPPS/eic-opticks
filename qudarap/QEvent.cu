#include "hip/hip_runtime.h"
#include <stdio.h>

#include "scuda.h"
#include "squad.h"
#include "srec.h"
#include "sphoton.h"
#include "sevent.h"

#include "iexpand.h"
#include "strided_range.h"
#include <thrust/device_vector.h>

/**
_QEvent_checkEvt
-----------------

Demonstrates using seed buffer to lookup genstep_id from photon_id 

**/

__global__ void _QEvent_checkEvt(sevent* evt, unsigned width, unsigned height)
{
    unsigned ix = blockIdx.x*blockDim.x + threadIdx.x;
    if( ix >= width ) return ;  

    unsigned photon_id = ix ; 
    unsigned genstep_id = evt->seed[photon_id] ; 
    const quad6& gs = evt->genstep[genstep_id] ; 
    int gencode = gs.q0.i.x ; 
    unsigned num_photon = evt->num_photon ; 

    printf("//_QEvent_checkEvt width %d height %d photon_id %3d genstep_id %3d  gs.q0.i ( %3d %3d %3d %3d )  gencode %d num_photon %d \n", 
       width,
       height,
       photon_id, 
       genstep_id, 
       gs.q0.i.x, 
       gs.q0.i.y,
       gs.q0.i.z, 
       gs.q0.i.w,
       gencode, 
       num_photon 
      );  
}

extern "C" void QEvent_checkEvt(dim3 numBlocks, dim3 threadsPerBlock, sevent* evt, unsigned width, unsigned height ) 
{
    printf("//QEvent_checkEvt width %d height %d \n", width, height );  
    _QEvent_checkEvt<<<numBlocks,threadsPerBlock>>>( evt, width, height  );
} 

/**
QEvent_count_genstep_photons
-------------------------------

NB this needs nvcc compilation due to the use of thrust but 
the method itself does not run on the device although the 
methods it invokes do run on the device. 

So the sevent* argument must be the CPU side instance 
which must be is holding GPU side pointers.

**/


//#ifdef DEBUG_QEVENT
struct printf_functor
{
    __host__ __device__ void operator()(int x){ printf("printf_functor %d\n", x); }
};
//#endif


/**
QEvent_count_genstep_photons
-----------------------------

Notice how using strided_range needs itemsize stride twice, 
because are grabbing single ints "numphoton" from each quad6 6*4 genstep 

**/


extern "C" unsigned QEvent_count_genstep_photons(sevent* evt)
{
    typedef typename thrust::device_vector<int>::iterator Iterator;

    thrust::device_ptr<int> t_gs = thrust::device_pointer_cast( (int*)evt->genstep ) ; 

#ifdef DEBUG_QEVENT
    printf("//QEvent_count_genstep_photons sevent::genstep_numphoton_offset %d  sevent::genstep_itemsize  %d  \n", 
            sevent::genstep_numphoton_offset, sevent::genstep_itemsize ); 
#endif

    strided_range<Iterator> gs_pho( 
        t_gs + sevent::genstep_numphoton_offset, 
        t_gs + evt->num_genstep*sevent::genstep_itemsize , 
        sevent::genstep_itemsize );    // begin, end, stride 

    evt->num_seed = thrust::reduce(gs_pho.begin(), gs_pho.end() );

#ifdef DEBUG_QEVENT
    //thrust::for_each( gs_pho.begin(), gs_pho.end(), printf_functor() );  
    printf("//QEvent_count_genstep_photons evt.num_genstep %d evt.num_seed %d evt.max_photon %d \n", evt->num_genstep, evt->num_seed, evt->max_photon ); 
#endif
    assert( evt->num_seed <= evt->max_photon ); 

    return evt->num_seed ; 
} 

/**
QEvent_fill_seed_buffer
-------------------------

Populates seed buffer using the numbers of photons per genstep from the genstep buffer.

See thrustrap/tests/iexpand_stridedTest.cu for the lead up to this

1. use GPU side genstep array to add the numbers of photons
   from each genstep giving the total number of photons and seeds *num_seeds*
   from all the gensteps

2. populate it by repeating genstep indices into it, 
   according to the number of photons in each genstep 
   
t_gs+sevent::genstep_numphoton_offset 
   q0.u.w of the quad6 genstep, which contains the number of photons 
   for this genstep


WARNING : SOMETHING HERE MESSES UP UNLESS THE SEED BUFFER IS ZEROED PRIOR TO THIS BEING CALLED

ACTUALLY THIS IS DUE TO A A LIMITATION OF IEXPAND, see sysrap/iexpand.h::

    NB the output device must be zeroed prior to calling iexpand. 
    This is because the iexpand is implemented ending with an inclusive_scan 
    to fill in the non-transition values which relies on initial zeroing.

**/

extern "C" void QEvent_fill_seed_buffer(sevent* evt )
{
#ifdef DEBUG_QEVENT
    printf("//QEvent_fill_seed_buffer evt.num_genstep %d evt.num_seed %d evt.max_photon %d \n", evt->num_genstep, evt->num_seed, evt->max_photon );      
#endif

    assert( evt->seed && evt->num_seed > 0 ); 
    assert( evt->num_seed <= evt->max_photon ); 

    thrust::device_ptr<int> t_seed = thrust::device_pointer_cast(evt->seed) ; 

    typedef typename thrust::device_vector<int>::iterator Iterator;

    thrust::device_ptr<int> t_gs = thrust::device_pointer_cast( (int*)evt->genstep ) ; 

    strided_range<Iterator> gs_pho( 
           t_gs + sevent::genstep_numphoton_offset, 
           t_gs + evt->num_genstep*sevent::genstep_itemsize, 
           sevent::genstep_itemsize );    // begin, end, stride 


    //thrust::for_each( gs_pho.begin(), gs_pho.end(), printf_functor() );  

    iexpand( gs_pho.begin(), gs_pho.end(), t_seed, t_seed + evt->num_seed );  

    //thrust::for_each( t_seed,  t_seed + evt->num_seed, printf_functor() );  

}



/**
QEvent_count_genstep_photons_and_fill_seed_buffer
---------------------------------------------------

This function does the same as the above two functions. 
It is invoked from QEvent::setGenstep

**/

extern "C" void QEvent_count_genstep_photons_and_fill_seed_buffer(sevent* evt )
{
    typedef typename thrust::device_vector<int>::iterator Iterator;

    thrust::device_ptr<int> t_gs = thrust::device_pointer_cast( (int*)evt->genstep ) ; 

#ifdef DEBUG_QEVENT
    printf("//QEvent_count_genstep_photons sevent::genstep_numphoton_offset %d  sevent::genstep_itemsize  %d  \n", 
            sevent::genstep_numphoton_offset, sevent::genstep_itemsize ); 
#endif


    strided_range<Iterator> gs_pho( 
        t_gs + sevent::genstep_numphoton_offset, 
        t_gs + evt->num_genstep*sevent::genstep_itemsize , 
        sevent::genstep_itemsize );    // begin, end, stride 

    evt->num_seed = thrust::reduce(gs_pho.begin(), gs_pho.end() );

#ifdef DEBUG_QEVENT
    printf("//QEvent_count_genstep_photons_and_fill_seed_buffer evt.num_genstep %d evt.num_seed %d evt.max_photon %d \n", evt->num_genstep, evt->num_seed, evt->max_photon );      
#endif

    bool expect_seed =  evt->seed && evt->num_seed > 0 ; 
    if(!expect_seed) printf("//QEvent_count_genstep_photons_and_fill_seed_buffer  evt.seed %s  evt.num_seed %d \n",  (evt->seed ? "YES" : "NO " ), evt->num_seed );  
    assert( expect_seed ); 

    bool num_seed_ok = evt->num_seed <= evt->max_photon ;

    if( num_seed_ok == false )
    {
        printf("//QEvent_count_genstep_photons_and_fill_seed_buffer FAIL evt.num_seed %d evt.max_photon %d num_seed_ok %d \n", evt->num_seed, evt->max_photon, num_seed_ok  ); 
    }

    assert( num_seed_ok ); 

    thrust::device_ptr<int> t_seed = thrust::device_pointer_cast(evt->seed) ; 

    //thrust::for_each( gs_pho.begin(), gs_pho.end(), printf_functor() );  

#ifdef DEBUG_QEVENT
    printf("//[QEvent_count_genstep_photons_and_fill_seed_buffer iexpand \n" );      
#endif


    iexpand( gs_pho.begin(), gs_pho.end(), t_seed, t_seed + evt->num_seed );  

    //thrust::for_each( t_seed,  t_seed + evt->num_seed, printf_functor() );  


#ifdef DEBUG_QEVENT
    printf("//]QEvent_count_genstep_photons_and_fill_seed_buffer iexpand \n" );      
#endif



}





#include "hip/hip_runtime.h"
#include <stdio.h>
#include "scuda.h"
#include "squad.h"
#include "qevent.h"

__global__ void _QEvent_checkEvt(qevent* evt, unsigned width, unsigned height)
{
    unsigned ix = blockIdx.x*blockDim.x + threadIdx.x;
    if( ix >= width ) return ;  

    unsigned photon_id = ix ; 
    unsigned genstep_id = evt->se[photon_id] ; 
    const quad6& gs = evt->gs[genstep_id] ; 
    int gencode = gs.q0.i.x ; 

    printf("//_QEvent_checkEvt width %d height %d photon_id %3d genstep_id %3d  gs.q0.i ( %3d %3d %3d %3d )  gencode %d \n", 
       width,
       height,
       photon_id, 
       genstep_id, 
       gs.q0.i.x, 
       gs.q0.i.y,
       gs.q0.i.z, 
       gs.q0.i.w,
       gencode 
      );  
}

extern "C" void QEvent_checkEvt(dim3 numBlocks, dim3 threadsPerBlock, qevent* evt, unsigned width, unsigned height ) 
{
    printf("//QEvent_checkEvt width %d height %d \n", width, height );  
    _QEvent_checkEvt<<<numBlocks,threadsPerBlock>>>( evt, width, height  );
} 



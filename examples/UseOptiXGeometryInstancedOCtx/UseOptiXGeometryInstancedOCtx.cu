#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019 Opticks Team. All Rights Reserved.
 *
 * This file is part of Opticks
 * (see https://bitbucket.org/simoncblyth/opticks).
 *
 * Licensed under the Apache License, Version 2.0 (the "License"); 
 * you may not use this file except in compliance with the License.  
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software 
 * distributed under the License is distributed on an "AS IS" BASIS, 
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.  
 * See the License for the specific language governing permissions and 
 * limitations under the License.
 */

#include <optix_world.h>
using namespace optix;

// from optixrap/cu/helpers.h

// Convert a float3 in [0,1)^3 to a uchar4 in [0,255]^4 -- 4th channel is set to 255
static __device__ __inline__ optix::uchar4 make_color(const optix::float3& c)
{
    return optix::make_uchar4( static_cast<unsigned char>(__saturatef(c.x)*255.99f),  // R 
                               static_cast<unsigned char>(__saturatef(c.y)*255.99f),  // G 
                               static_cast<unsigned char>(__saturatef(c.z)*255.99f),  // B 
                               255u);                                                 // A 
}

/*
static __device__ __inline__ optix::uchar4 make_color(const optix::float4& c)
{
    return optix::make_uchar4( static_cast<unsigned char>(__saturatef(c.x)*255.99f),   // R 
                               static_cast<unsigned char>(__saturatef(c.y)*255.99f),   // G
                               static_cast<unsigned char>(__saturatef(c.z)*255.99f),   // B 
                               static_cast<unsigned char>(__saturatef(c.w)*255.99f));  // A
}
*/


struct PerRayData
{
    float3 result;
    uint4  inid ;  
    float4 post ; 
    float4 posi ; 
};


rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(unsigned,     radiance_ray_type, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

rtDeclareVariable(rtObject,      top_object, , );

rtBuffer<uchar4, 2>   output_buffer;
rtBuffer<uint4, 2>    inid_buffer;
rtBuffer<float4, 2>   post_buffer;
rtBuffer<float4, 2>   posi_buffer;


// from geometry intersect 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, );  
rtDeclareVariable(uint4, intersect_identity,   attribute intersect_identity, );  
rtDeclareVariable(unsigned, intersect_id,   attribute intersect_id, );  

rtDeclareVariable(PerRayData, prd, rtPayload, );


rtDeclareVariable(optix::Ray,           raycur, rtCurrentRay, );
rtDeclareVariable(float,                  t, rtIntersectionDistance, );

rtDeclareVariable(int,   texture_id, , );


RT_PROGRAM void raygen_texture_test()
{
    float2 d = make_float2(launch_index) / make_float2(launch_dim) ;  // 0->1

    //output_buffer[launch_index] = rtTex2DLayered<uchar4>( texture_id, d.x, d.y, layer );
    output_buffer[launch_index] = rtTex2D<uchar4>( texture_id, d.x, d.y );
    //output_buffer[launch_index] = make_uchar4( 255, 0, 0, 255 ); 
}

RT_PROGRAM void raygen()
{
    PerRayData prd;
    prd.result = make_float3( 1.f, 0.f, 0.f ) ;
    prd.inid = make_uint4(0,0,0,0); 
    prd.post = make_float4( 0.f, 0.f, 0.f, 0.f ) ;
    prd.posi = make_float4( 0.f, 0.f, 0.f, 0.f ) ;

    float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f ;   // -1:1

    optix::Ray ray = optix::make_Ray( eye, normalize(d.x*U + d.y*V + W), radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX) ; 
    rtTrace(top_object, ray, prd);

     //rtPrintf("//raygen launch_index.x %u launch_index.y %u launch_dim.x %u launch_dim.y %u \n", launch_index.x , launch_index.y, launch_dim.x , launch_dim.y   );
    output_buffer[launch_index] = make_color( prd.result ) ; 
    // make_uchar4(  255u, 0u, 0u,255u) ;  // red  (was expecting BGRA get RGBA)

    inid_buffer[launch_index] = prd.inid ; 
    post_buffer[launch_index] = prd.post ; 
    posi_buffer[launch_index] = prd.posi ; 
}

// Returns shading normal as the surface shading result
RT_PROGRAM void closest_hit_local()
{
    float3 isect = raycur.origin + t*raycur.direction ; 
    const float3 local = rtTransformPoint( RT_WORLD_TO_OBJECT, isect );  
    prd.result = normalize(local)*0.5f + 0.5f ; 
    prd.inid = intersect_identity ; 
    prd.post = make_float4( isect, t ); 
    prd.posi = make_float4( isect, __uint_as_float(intersect_id) ); 
}
RT_PROGRAM void closest_hit_global()
{
    float3 isect = raycur.origin + t*raycur.direction ; 
    prd.result = normalize(isect)*0.5f + 0.5f ;    // coloring clearly global like this
    prd.inid = intersect_identity ; 
    prd.post = make_float4( isect, t ); 
    prd.posi = make_float4( isect, __uint_as_float(intersect_id) ); 
}
RT_PROGRAM void closest_hit_normal()
{
    float3 isect = raycur.origin + t*raycur.direction ; 
    prd.result = normalize(rtTransformNormal(RT_WORLD_TO_OBJECT, shading_normal))*0.5f + 0.5f;
    prd.inid = intersect_identity ; 
    prd.post = make_float4( isect, t ); 
    prd.posi = make_float4( isect, __uint_as_float(intersect_id) ); 
    //rtPrintf("//closest_hit_normal intersect_id %d \n", intersect_id); 
}
RT_PROGRAM void closest_hit_textured()
{
    float3 isect = raycur.origin + t*raycur.direction ; 
    const float3 local = rtTransformPoint( RT_WORLD_TO_OBJECT, isect );  
    const float3 norm = normalize(local) ;  

    float f_theta = acos( norm.z )/M_PIf;                 // polar 0->pi ->  0->1
    float f_phi_ = atan2( norm.y, norm.x )/(2.f*M_PIf) ;  // azimuthal 0->2pi ->  0->1
    float f_phi = f_phi_ > 0.f ? f_phi_ : f_phi_ + 1.f ;  //  

    uchar4 val = rtTex2D<uchar4>( texture_id, f_phi, f_theta );
    float3 result = make_float3( float(val.x)/255.99f,  float(val.y)/255.99f,  float(val.z)/255.99f ) ;   

    prd.result = result ;  ; 
    prd.inid = intersect_identity ; 
    prd.post = make_float4( isect, t ); 
    prd.posi = make_float4( isect, __uint_as_float(intersect_id) ); 
}
RT_PROGRAM void miss()
{
    prd.result = make_float3(1.f, 1.f, 1.f) ;
    prd.inid = make_uint4( 0,0,0,0)  ; 
    prd.post = make_float4(0.f,0.f,0.f,0.f); 
    prd.posi = make_float4(0.f,0.f,0.f, __uint_as_float(0u)); 
}





RT_PROGRAM void printTest0()
{
    unsigned long long index = launch_index.x ;
    rtPrintf("//printTest0 d:%d launch_index.x %u launch_index.y %u launch_dim.x %u launch_dim.y %u \n", index, launch_index.x , launch_index.y, launch_dim.x , launch_dim.y   );
}
RT_PROGRAM void printTest1()
{
    unsigned long long index = launch_index.x ;
    rtPrintf("//printTest1 llu:%llu launch_index.x %u launch_index.y %u launch_dim.x %u launch_dim.y %u \n", index, launch_index.x , launch_index.y, launch_dim.x , launch_dim.y   );
}
RT_PROGRAM void exception()
{
    rtPrintExceptionDetails();
}



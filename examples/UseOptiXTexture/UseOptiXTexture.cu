#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019 Opticks Team. All Rights Reserved.
 *
 * This file is part of Opticks
 * (see https://bitbucket.org/simoncblyth/opticks).
 *
 * Licensed under the Apache License, Version 2.0 (the "License"); 
 * you may not use this file except in compliance with the License.  
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software 
 * distributed under the License is distributed on an "AS IS" BASIS, 
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.  
 * See the License for the specific language governing permissions and 
 * limitations under the License.
 */

#include "UseOptiXTexture.h"
#include <optix_world.h>
using namespace optix;

rtBuffer<float,3> tex_buffer ; 
rtBuffer<float,3> out_buffer ; 


rtTextureSampler<float, 3> tex_sampler ;

rtDeclareVariable(uint3, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint3, launch_dim,   rtLaunchDim, );



RT_PROGRAM void readWrite()
{
    unsigned x = launch_index.x ; 
    unsigned y = launch_index.y ; 
    unsigned z = launch_index.z ; 
    unsigned nx = launch_dim.x ; 
    unsigned ny = launch_dim.y ; 
    unsigned nz = launch_dim.z ; 

#ifdef FROM_BUF
    float val = tex_buffer[launch_index] ;  
#else
    float3 tex_coord = make_float3( float(x), float(y), float(z)); 
    float val = tex3D( tex_sampler, tex_coord.x, tex_coord.y, tex_coord.z );
#endif

    rtPrintf("//UseOptiXTexture.cu:readWrite launch_index.xyz ( %u %u %u ) launch_dim.xyz (%u %u %u )  val %10.3f \n", 
         x, 
         y, 
         z, 
         nx, 
         ny, 
         nz, 
         val
       );

    out_buffer[launch_index] = val ; 
}

RT_PROGRAM void exception()
{
    rtPrintExceptionDetails();
}



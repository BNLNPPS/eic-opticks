// nvcc compute_capability.cu -o /tmp/compute_capability 

#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int, char**)
{
    unsigned dev = 0 ; 
    hipDeviceProp_t p;
    hipGetDeviceProperties(&p, dev);
    printf("%d%d\n", p.major, p.minor);
}

#include "hip/hip_runtime.h"

__global__ void CRng_generate(int threads_per_launch, hiprandState* rng_states, float* d_arr )
{
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id >= threads_per_launch) return;

    float u = hiprand_uniform(&rng_states[id]); 
    d_arr[id] = u ;   
}


